#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <cstdlib>

#include "FlyingEdgesAlgorithm.h"

#include "../util/LoadImage.h"
#include "../util/SaveTriangleMesh.h"

#include "../util/Timer.h"
#include "../mantevoCommon/YAML_Doc.hpp"

__global__ void doubler(scalar_t* a)
{
    int i = blockIdx.x;

    a[i] = a[i]*2;
}

int main(int argc, char* argv[])
{
/*
    std::vector<scalar_t> vec(10);
    scalar_t c = 9.3;
    for(scalar_t& s: vec)
    {
        s = c;
        c += 1.15;
    }

    scalar_t* deviceVec;
    hipMalloc(&deviceVec, 10*sizeof(scalar_t));

    hipMemcpy(deviceVec, vec.data(), 10*sizeof(scalar_t), hipMemcpyHostToDevice);

    doubler<<<10,1>>>(deviceVec);

    hipMemcpy(vec.data(), deviceVec, 10*sizeof(scalar_t), hipMemcpyDeviceToHost);

    hipFree(deviceVec);

    for(scalar_t const& s: vec)
        std::cout << s << std::endl;

    return 0;
*/

    float isoval;
    bool isovalSet = false;
    char* vtkFile = NULL;
    char* outFile = NULL;
    std::string yamlDirectory = "";
    std::string yamlFileName  = "";

    // Read command line arguments
    for(int i=0; i<argc; i++)
    {
        if( (strcmp(argv[i], "-i") == 0) || (strcmp(argv[i], "-input_file") == 0))
        {
            vtkFile = argv[++i];
        }
        else if( (strcmp(argv[i], "-o") == 0) || (strcmp(argv[i], "-output_file") == 0))
        {
            outFile = argv[++i];
        }
        else if( (strcmp(argv[i], "-v") == 0) || (strcmp(argv[i], "-isoval") == 0))
        {
            isovalSet = true;
            isoval = atof(argv[++i]);
        }
        else if( (strcmp(argv[i], "-y") == 0) || (strcmp(argv[i], "-yaml_output_file") == 0))
        {
            std::string wholeFile(argv[++i]);

            std::size_t pos = wholeFile.rfind("/");
            if(pos == std::string::npos)
            {
                yamlDirectory = "./";
                yamlFileName = wholeFile;
            }
            else
            {
                yamlDirectory = wholeFile.substr(0, pos + 1);
                yamlFileName = wholeFile.substr(pos + 1);
            }
        }
        else if( (strcmp(argv[i], "-h") == 0) || (strcmp(argv[i], "-help") == 0))
        {
            std::cout <<
                "Serial Flying Edges Options:"    << std::endl <<
                "  -input_file (-i)"              << std::endl <<
                "  -output_file (-o)"             << std::endl <<
                "  -isoval (-v)"                  << std::endl <<
                "  -yaml_output_file (-y)"        << std::endl <<
                "  -help (-h)"                    << std::endl;
            return 0;
        }
    }

    if(isovalSet == false || vtkFile == NULL || outFile == NULL)
    {
        std::cout << "Error: isoval, input_file and output_file must be set." << std::endl <<
                     "Try -help" << std::endl;
        return 0;
    }

    YAML_Doc doc("Flying Edges", "0.1", yamlDirectory, yamlFileName);

    doc.add("Flying Edges Algorithm", "cuda");
    doc.add("Volume image data file path", vtkFile);
    doc.add("Polygonal mesh output file", outFile);
    doc.add("Isoval", isoval);

    util::Image3D image = util::loadImage(vtkFile);

    doc.add("File x-dimension", image.xdimension());
    doc.add("File y-dimension", image.ydimension());
    doc.add("File z-dimension", image.zdimension());

    util::Timer runTime;

    util::Timer toDeviceTime;
    FlyingEdgesAlgorithm algo(image, isoval);
    toDeviceTime.stop();

    util::Timer runTimePass1;
    algo.pass1();
    runTimePass1.stop();

    util::Timer runTimePass2;
    algo.pass2();
    runTimePass2.stop();

    util::Timer runTimePass3;
    algo.pass3();
    runTimePass3.stop();

    util::Timer runTimePass4;
//    algo.pass4();
    runTimePass4.stop();

    util::Timer fromDeviceTime;
    util::TriangleMesh mesh = algo.moveOutput();
    fromDeviceTime.stop();

    runTime.stop();

    doc.add("Number of vertices in mesh", mesh.numberOfVertices());
    doc.add("Number of triangles in mesh", mesh.numberOfTriangles());

    doc.add("To Device", "");
    doc.get("To Device")->add("CPU Time (clicks)", toDeviceTime.getTotalTicks());
    doc.get("To Device")->add("CPU Time (seconds)", toDeviceTime.getCPUtime());
    doc.get("To Device")->add("Wall Time (seconds)", toDeviceTime.getWallTime());

    doc.add("Pass 1", "");
    doc.get("Pass 1")->add("CPU Time (clicks)", runTimePass1.getTotalTicks());
    doc.get("Pass 1")->add("CPU Time (seconds)", runTimePass1.getCPUtime());
    doc.get("Pass 1")->add("Wall Time (seconds)", runTimePass1.getWallTime());

    doc.add("Pass 2", "");
    doc.get("Pass 2")->add("CPU Time (clicks)", runTimePass2.getTotalTicks());
    doc.get("Pass 2")->add("CPU Time (seconds)", runTimePass2.getCPUtime());
    doc.get("Pass 2")->add("Wall Time (seconds)", runTimePass2.getWallTime());

    doc.add("Pass 3", "");
    doc.get("Pass 3")->add("CPU Time (clicks)", runTimePass3.getTotalTicks());
    doc.get("Pass 3")->add("CPU Time (seconds)", runTimePass3.getCPUtime());
    doc.get("Pass 3")->add("Wall Time (seconds)", runTimePass3.getWallTime());

    doc.add("Pass 4", "");
    doc.get("Pass 4")->add("CPU Time (clicks)", runTimePass4.getTotalTicks());
    doc.get("Pass 4")->add("CPU Time (seconds)", runTimePass4.getCPUtime());
    doc.get("Pass 4")->add("Wall Time (seconds)", runTimePass4.getWallTime());

    doc.add("From Device", "");
    doc.get("From Device")->add("CPU Time (clicks)", fromDeviceTime.getTotalTicks());
    doc.get("From Device")->add("CPU Time (seconds)", fromDeviceTime.getCPUtime());
    doc.get("From Device")->add("Wall Time (seconds)", fromDeviceTime.getWallTime());

    doc.add("Total Program CPU Time (clicks)", runTime.getTotalTicks());
    doc.add("Total Program CPU Time (seconds)", runTime.getCPUtime());
    doc.add("Total Program WALL Time (seconds)", runTime.getWallTime());

    std::cout << doc.generateYAML();

//    util::saveTriangleMesh(mesh, outFile);
}
