#include "hip/hip_runtime.h"
/*
 * flyingEdgesAlgorithm.cpp
 *
 *  Created on: Feb 17, 2017
 *      Author: dbourge
 */
#include "FlyingEdgesAlgorithm.h"

#include "CudaMarchingCubesTables.h"

#include <numeric>

#include <iostream> // TODO

// TODO make sure pointValues stored in const memory

///////////////////////////////////////////////////////////////////////////////
// Pass 1 of the algorithm
///////////////////////////////////////////////////////////////////////////////

__device__
uchar calcCaseEdge(
    bool const& prevEdge,
    bool const& currEdge)
{
    // o -- is greater than or equal to
    // case 0: (i-1) o-----o (i) | (_,j,k)
    // case 1: (i-1) x-----o (i) | (_,j+1,k)
    // case 2: (i-1) o-----x (i) | (_,j,k+1)
    // case 3: (i-1) x-----x (i) | (_,j+1,k+1)
    if(prevEdge && currEdge)
        return 0;
    if(!prevEdge && currEdge)
        return 1;
    if(prevEdge && !currEdge)
        return 2;
    else // !prevEdge && !currEdge
        return 3;
}

__global__
void pass1gpu_edgeCases(
    scalar_t* pointValues,
    scalar_t isoval,
    int nx, int ny,
    uchar* edgeCases)
{
    // (nx-1, ny, nz) > comes as (nx-1, ny*nz)
    // Each row has several blocks
    // Each thread is one point

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y % ny;
    int k = blockIdx.y / ny;

    __shared__ bool isGE[FE_BLOCK_WIDTH_PLUS_ONE];

    if(i < nx)
        isGE[threadIdx.x] = pointValues[k*nx*ny + j*nx + i] >= isoval;

    if(threadIdx.x == 0 && i < nx-1)
    {
        isGE[blockDim.x] = pointValues[k*nx*ny + j*nx + i + blockDim.x] >= isoval;
    }

    __syncthreads();

    if(i < nx-1)
    {
        uchar caseEdge = calcCaseEdge(isGE[threadIdx.x], isGE[threadIdx.x + 1]);
        edgeCases[k*(nx-1)*ny + j*(nx-1) + i] = caseEdge;
    }
}

__global__
void pass1gpu_trim(
    int nx, int ny, int nz,                    // input
    uchar* edgeCases,                          // input
    FlyingEdgesAlgorithm::gridEdge* gridEdges) // output
{
    // (1, ny, nz) > comes as (ny, nz)

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if(j >= ny || k >= nz)
        return;

    size_t xl = nx;
    size_t xr = 0;

    uchar* curEdgeCases = edgeCases + k*(nx-1)*ny + j*(nx-1);

    for(int i = 0; i != nx-1; ++i)
    {
        if(curEdgeCases[i] == 1 || curEdgeCases[i] == 2)
        {
            if(xl == nx)
                xl = i;
            xr = i+1;
        }
    }

    gridEdges[k*ny + j].xl = xl;
    gridEdges[k*ny + j].xr = xr;
}

void FlyingEdgesAlgorithm::pass1()
{
    int tx = FE_BLOCK_WIDTH;
    uint3 gridDim = make_uint3(((nx-1) + tx - 1) / tx, ny*nz, 1);
    uint3 blockDim = make_uint3(tx, 1, 1);
    pass1gpu_edgeCases<<<gridDim, blockDim>>>(
        pointValues,
        isoval,
        nx, ny,
        edgeCases);

    int ty = FE_BLOCK_WIDTH_Y;
    int tz = FE_BLOCK_WIDTH_Z;
    gridDim = make_uint3((ny + ty - 1) / ty, (nz + tz - 1) / tz, 1);
    blockDim = make_uint3(ty, tz, 1);

    pass1gpu_trim<<<gridDim, blockDim>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges);

    hipDeviceSynchronize();

    /////////////////////////////////////
    // WHAT IS GOING ON?..It works now //  TODO
    /////////////////////////////////////

    int numGE = nz*ny;
    gridEdge* hostGEs = (gridEdge*)malloc(numGE*sizeof(gridEdge));
    hipMemcpy(hostGEs, gridEdges, numGE*sizeof(gridEdge),
               hipMemcpyDeviceToHost);

    size_t countL = 0;
    size_t countR = 0;
    for(int idx = 0; idx != numGE; ++idx)
    {
        countL += hostGEs[idx].xl;
        countR += hostGEs[idx].xr;
    }

    std::cout << "xl, xr: " << countL << ", " << countR << std::endl;

    free(hostGEs);
}

/*
void FlyingEdgesAlgorithm::pass1()
{
    // For each (j, k):
    //  - for each edge i along fixed (j, k) gridEdge, fill edgeCases with
    //    cut information.
    //  - find the locations for computational trimming, xl and xr
    for(size_t k = 0; k != nz; ++k) {
    for(size_t j = 0; j != ny; ++j)
    {
        auto curEdgeCases = edgeCases.begin() + (nx-1) * (k*ny + j);
        auto curPointValues = image.getRowIter(j, k);

        gridEdge& curGridEdge = gridEdges[k*ny + j];

        std::array<bool, 2> isGE;
        isGE[0] = (curPointValues[0] >= isoval);
        for(int i = 1; i != nx; ++i)
        {
            isGE[i%2] = (curPointValues[i] >= isoval);

            curEdgeCases[i-1] = calcCaseEdge(isGE[(i+1)%2], isGE[i%2]);

            // If the edge is cut
            if(curEdgeCases[i-1] == 1 || curEdgeCases[i-1] == 2)
            {
                if(curGridEdge.xl == 0)
                    curGridEdge.xl == i-1;

                curGridEdge.xr = i;
            }
        }
    }}
}

///////////////////////////////////////////////////////////////////////////////
*/
///////////////////////////////////////////////////////////////////////////////
// Pass 2 of the algorithm
///////////////////////////////////////////////////////////////////////////////

__device__
void calcTrimValues(
    int& xl, int& xr,
    FlyingEdgesAlgorithm::gridEdge const& ge0,
    FlyingEdgesAlgorithm::gridEdge const& ge1,
    FlyingEdgesAlgorithm::gridEdge const& ge2,
    FlyingEdgesAlgorithm::gridEdge const& ge3)
{
    xl = min(ge0.xl, min(ge1.xl, min(ge2.xl, ge3.xl)));
    xr = max(ge0.xr, max(ge1.xr, max(ge2.xr, ge3.xr)));
}

__device__
uchar calcCubeCase(
    uchar const& ec0, uchar const& ec1,
    uchar const& ec2, uchar const& ec3)
{
    // ec0 | (_,j,k)
    // ec1 | (_,j+1,k)
    // ec2 | (_,j,k+1)
    // ec3 | (_,j+1,k+1)

    uchar caseId = 0;
    if((ec0 == 0) || (ec0 == 2)) // 0 | (i,j,k)
        caseId |= 1;
    if((ec0 == 0) || (ec0 == 1)) // 1 | (i+1,j,k)
        caseId |= 2;
    if((ec1 == 0) || (ec1 == 1)) // 2 | (i+1,j+1,k)
        caseId |= 4;
    if((ec1 == 0) || (ec1 == 2)) // 3 | (i,j+1,k)
        caseId |= 8;
    if((ec2 == 0) || (ec2 == 2)) // 4 | (i,j,k+1)
        caseId |= 16;
    if((ec2 == 0) || (ec2 == 1)) // 5 | (i+1,j,k+1)
        caseId |= 32;
    if((ec3 == 0) || (ec3 == 1)) // 6 | (i+1,j+1,k+1)
        caseId |= 64;
    if((ec3 == 0) || (ec3 == 2)) // 7 | (i,j+1,k+1)
        caseId |= 128;
    return caseId;
}

__global__
void pass2gpu_cubeCases(
    int nx, int ny, int nz,
    uchar* edgeCases,
    FlyingEdgesAlgorithm::gridEdge* gridEdges,
    int* triCounter,
    uchar* cubeCases)
{
    // (1, ny-1, nz-1) > comes as (ny-1, nz-1)
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if(j >= ny-1 || k >= nz-1)
        return;

    FlyingEdgesAlgorithm::gridEdge& ge0 = gridEdges[k*ny + j];
    FlyingEdgesAlgorithm::gridEdge& ge1 = gridEdges[k*ny + j + 1];
    FlyingEdgesAlgorithm::gridEdge& ge2 = gridEdges[(k+1)*ny + j];
    FlyingEdgesAlgorithm::gridEdge& ge3 = gridEdges[(k+1)*ny + j + 1];

    uchar* ec0 = edgeCases + k*ny*(nx-1) + j*(nx-1); // (nx-1)*(k*ny + j);
    uchar* ec1 = edgeCases + k*ny*(nx-1) + (j+1)*(nx-1); // (nx-1)*(k*ny + j + 1);
    uchar* ec2 = edgeCases + (k+1)*ny*(nx-1) + j*(nx-1); // (nx-1)*((k+1)*ny + j);
    uchar* ec3 = edgeCases + (k+1)*ny*(nx-1) + (j+1)*(nx-1); // (nx-1)*((k+1)*ny + j + 1);

    int xl, xr;
    calcTrimValues(xl, xr, ge0, ge1, ge2, ge3);

    int triCount = 0;
    uchar* curCubeCases = cubeCases + k*(nx-1)*(ny-1) + j*(nx-1);

    int xstart = 0;
    int ystart = 0;
    int zstart = 0; // TODO don't set initial values in gridEdge Constructor;

    const bool* isCut;
    for(int i = xl; i != xr; ++i) // What happens here on a gpu?
                                  // I imagine it takes the max xr-xl of all blocks
    {
        // TODO why is this needed?
        if (i >= nx-1)
            return;

        uchar caseId = calcCubeCase(ec0[i], ec1[i], ec2[i], ec3[i]);

        curCubeCases[i] = caseId; // THIS LINE BREAKS EVERYTHING

        // Can't imagine this would do anything on a gpu unless all threads
        // on a block evaluated to the same value.
        if(caseId == 0 || caseId == 255)
        {
            continue;
        }

        triCount += cuda_util::numTris[caseId];
        isCut = cuda_util::isCut[caseId]; // if xr == nx-1, then xr-1 is cut
                                          // so this will be set

        xstart += isCut[0];
        ystart += isCut[3];
        zstart += isCut[8];
    }

    triCounter[k*(ny-1) + j] = triCount;

    if(xr == nx-1)
    {
        // isCut was set at i = xr-1
        ystart += isCut[1];
        zstart += isCut[9];
    }

    ge0.xstart = xstart;
    ge0.ystart = ystart;
    ge0.zstart = zstart;
}

__global__
void pass2gpu_ghost_xz(
    int nx, int ny, int nz,
    uchar* edgeCases,
    FlyingEdgesAlgorithm::gridEdge* gridEdges)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    //if(k >= nz-1)
    //    return;
    if(k >= nz) // This function will deal with gridEdge at (_, ny-1, nz-1)
        return;

    bool isCorner = k == nz-1;

    int j = ny-1;

    FlyingEdgesAlgorithm::gridEdge& ge0 = gridEdges[k*ny + j];
    // If isCorner, this is just bogus.
    FlyingEdgesAlgorithm::gridEdge& ge1 = gridEdges[(1-isCorner)*(k+1)*ny + j];

    uchar* ec0 = edgeCases + k*ny*(nx-1) + j*(nx-1);
    // If isCorner, this is just bogus
    uchar* ec1 = edgeCases + (1-isCorner)*(k+1)*ny*(nx-1) + j*(nx-1);

    int xl = min(ge0.xl, nx*isCorner + (1-isCorner)*ge1.xl);
    int xr = max(ge0.xr, (1-isCorner)*ge1.xr);

    if(xl >= xr)
        return;

    int xstart = 0;
    int zstart = 0; // TODO don't set initial values in gridEdge Constructor;

    uchar c0;
    uchar c1;

    for(int i = xl; i != xr; ++i)
    {
        c0 = ec0[i];
        c1 = ec1[i];

        // see if the edges are cut
        xstart += (c0 == 1 || c0 == 2);

        // bogus if isCorner
        zstart += ( (c0 == 0 && c1 == 1) || (c0 == 0 && c1 == 3) ||
                    (c0 == 1 && c1 == 2) || (c0 == 2 && c1 == 3) );
    }

    if(xr == nx-1)
    {
        // bogus if isCorner
        zstart += ( (c0 == 0 && c1 == 2) || (c0 == 0 && c1 == 3) ||
                    (c0 == 1 && c1 == 2) || (c0 == 1 && c1 == 3) );
    }

    ge0.xstart = xstart;
    ge0.ystart = 0;
    ge0.zstart = zstart*(1-isCorner);
}

__global__
void pass2gpu_ghost_xy(
    int nx, int ny, int nz,
    uchar* edgeCases,
    FlyingEdgesAlgorithm::gridEdge* gridEdges)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(j >= ny-1)
        return;

    int k = nz-1;

    FlyingEdgesAlgorithm::gridEdge& ge0 = gridEdges[k*ny + j];
    FlyingEdgesAlgorithm::gridEdge& ge1 = gridEdges[k*ny + j + 1];

    uchar* ec0 = edgeCases + k*ny*(nx-1) + j*(nx-1);
    uchar* ec1 = edgeCases + k*ny*(nx-1) + (j+1)*(nx-1);

    int xl = min(ge0.xl, ge1.xl);
    int xr = max(ge0.xr, ge1.xr);

    if(xl >= xr)
        return;

    int xstart = 0;
    int ystart = 0; // TODO don't set initial values in gridEdge Constructor;

    uchar c0;
    uchar c1;

    for(int i = xl; i != xr; ++i)
    {
        c0 = ec0[i];
        c1 = ec1[i];

        // see if the edges are cut
        xstart += (c0 == 1 || c0 == 2);
        ystart += ( (c0 == 0 && c1 == 1) || (c0 == 0 && c1 == 3) ||
                    (c0 == 1 && c1 == 2) || (c0 == 2 && c1 == 3) );
    }

    if(xr == nx-1)
    {
        ystart += ( (c0 == 0 && c1 == 2) || (c0 == 0 && c1 == 3) ||
                    (c0 == 1 && c1 == 2) || (c0 == 1 && c1 == 3) );
    }

    ge0.xstart = xstart;
    ge0.ystart = ystart;
    ge0.zstart = 0;
}

// TOO SLOW! done in xz ghost function
//__global__
//void pass2gpu_ghost_xyz(
//    int nx, int ny, int nz,
//    uchar* edgeCases,
//    FlyingEdgesAlgorithm::gridEdge* gridEdges)
//{
//    int j = ny-1;
//    int k = nz-1;
//
//    FlyingEdgesAlgorithm::gridEdge& ge = gridEdges[k*ny + j];
//    uchar* ec = edgeCases + k*ny*(nx-1) + j*(nx-1);
//
//    int xl = ge.xl;
//    int xr = ge.xr;
//
//    int xstart = 0;
//
//    uchar c;
//
//    for(int i = xl; i != xr; ++i)
//    {
//        c = ec[i];
//        xstart += (c == 1 || c == 2);
//    }
//
//    ge.xstart = xstart;
//    ge.ystart = 0;
//    ge.zstart = 0;
//}

void FlyingEdgesAlgorithm::pass2()
{
    // pass2 calculates
    //   1) cubeCases for each block ray
    //   2) triCount for each block ray
    //   3) edgeRay count

    // 1st kernel: Calculate the 0, 1, 2 edge ray, cube cases, tricount
    // 2nd kernel: Calculate lost edges

    int ty = FE_BLOCK_WIDTH_Y;
    int tz = FE_BLOCK_WIDTH_Z;
    uint3 gridDim = make_uint3(((ny-1) + ty - 1) / ty, ((nz-1) + tz - 1) / tz, 1);
    uint3 blockDim = make_uint3(ty, tz, 1);

    pass2gpu_cubeCases<<<gridDim, blockDim>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges,   // modified
        triCounter,  // modified
        cubeCases);  // modified
    ////////////////////////////////////////////////////////////////////////////////////////////////////
    size_t sz = (nx-1)*(ny-1)*(nz-1)*sizeof(uchar);
    hipDeviceSynchronize();
    uchar* hostCubeCases = (uchar*)malloc(sz);
    hipMemcpy(hostCubeCases, cubeCases,
               sz, hipMemcpyDeviceToHost);

    int count = 0;
    // TODO hostCubeCases is not the same every time.
    for(int i = 0; i != (nx-1)*(ny-1)*(nz-1); ++i)
    {
        if(hostCubeCases[i] != 0 && hostCubeCases[i] != 255)
            count += hostCubeCases[i];
    }
    std::cout << "Count cube cases " << count << std::endl;

    free(hostCubeCases);
    ////////////////////////////////////////////////////////////////////////////////////////////////////

    // TODO these can be launched and executed independently of each other
    int bw = FE_BLOCK_WIDTH;

    // Making sure that the xz face takes care of the (_, ny-1, nz-1) gridEdge
    // BE CAREFUL. xz takes care of corner. don't use (nz-1)
    pass2gpu_ghost_xz<<<(nz + bw - 1) / bw, bw>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges);
    pass2gpu_ghost_xy<<<((ny-1) + bw - 1) / bw, bw>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges);

    hipDeviceSynchronize();

// This is prohibitively slow so pass2gpu_ghost_xz covers it now
//    pass2gpu_ghost_xyz<<<1, 1, 3>>>(
//        nx, ny, nz,
//        edgeCases,
//        gridEdges);
}

/*
void FlyingEdgesAlgorithm::pass2()
{
    // For each (j, k):
    //  - for each cube (i, j, k) calculate caseId and number of gridEdge cuts
    //    in the x, y and z direction.
    for(size_t k = 0; k != nz-1; ++k) {
    for(size_t j = 0; j != ny-1; ++j)
    {
        // find adjusted trim values
        size_t xl, xr;
        calcTrimValues(xl, xr, j, k); // xl, xr set in this function

        // ge0 is owned by this (i, j, k). ge1, ge2 and ge3 are only used for
        // boundary cells.
        gridEdge& ge0 = gridEdges[k*ny + j];
        gridEdge& ge1 = gridEdges[k*ny + j + 1];
        gridEdge& ge2 = gridEdges[(k+1)*ny + j];
        gridEdge& ge3 = gridEdges[(k+1)*ny + j + 1];

        // ec0, ec1, ec2 and ec3 were set in pass 2. They are used
        // to calculate the cell caseId.
        auto const& ec0 = edgeCases.begin() + (nx-1)*(k*ny + j);
        auto const& ec1 = edgeCases.begin() + (nx-1)*(k*ny + j + 1);
        auto const& ec2 = edgeCases.begin() + (nx-1)*((k+1)*ny + j);
        auto const& ec3 = edgeCases.begin() + (nx-1)*((k+1)*ny + j + 1);

        // Count the number of triangles along this row of cubes.
        size_t& curTriCounter = *(triCounter.begin() + k*(ny-1) + j);

        auto curCubeCaseIds = cubeCases.begin() + (nx-1)*(k*(ny-1) + j);

        bool isYEnd = (j == ny-2);
        bool isZEnd = (k == nz-2);

        for(size_t i = xl; i != xr; ++i)
        {
            bool isXEnd = (i == nx-2);

            // using edgeCases from pass 2, compute cubeCases for this cube
            uchar caseId = calcCubeCase(ec0[i], ec1[i], ec2[i], ec3[i]);

            curCubeCaseIds[i] = caseId;

            // If the cube has no triangles through it
            if(caseId == 0 || caseId == 255)
            {
                continue;
            }

            curTriCounter += util::numTris[caseId];

            const bool* isCut = util::isCut[caseId]; // size 12

            ge0.xstart += isCut[0];
            ge0.ystart += isCut[3];
            ge0.zstart += isCut[8];

            // Note: Each 'gridCell' contains four gridEdges running along it,
            //       ge0, ge1, ge2 and ge3. Each gridCell can access it's own
            //       ge0 but ge1, ge2 and ge3 are owned by other gridCells.
            //       Accessing ge1, ge2 and ge3 leads to a race condition
            //       unless gridCell is along the boundry of the image.
            //
            //       To really make sense of the indices, it helps to draw
            //       out the following picture of a cube with the appropriate
            //       labels:
            //         v0 is at (i,   j,   k)
            //         v1       (i+1, j,   k)
            //         v2       (i+1, j+1, k)
            //         v3       (i,   j+1, k)
            //         v4       (i,   j,   k+1)
            //         v5       (i+1, j,   k+1)
            //         v6       (i+1, j+1, k+1)
            //         v7       (i,   j+1, k+1)
            //         e0  connects v0 to v1 and is parallel to the x-axis
            //         e1           v1    v2                        y
            //         e2           v2    v3                        x
            //         e3           v0    v3                        y
            //         e4           v4    v5                        x
            //         e5           v5    v6                        y
            //         e6           v6    v7                        x
            //         e7           v4    v7                        y
            //         e8           v0    v4                        z
            //         e9           v1    v5                        z
            //         e10          v3    v7                        z
            //         e11          v2    v6                        z

            // Handle cubes along the edge of the image
            if(isXEnd)
            {
                ge0.ystart += isCut[1];
                ge0.zstart += isCut[9];
            }
            if(isYEnd)
            {
                ge1.xstart += isCut[2];
                ge1.zstart += isCut[10];
            }
            if(isZEnd)
            {
                ge2.xstart += isCut[4];
                ge2.ystart += isCut[7];
            }

            if(isXEnd and isYEnd)
            {
                ge1.zstart += isCut[11];
            }
            if(isXEnd and isZEnd)
            {
                ge2.ystart += isCut[5];
            }
            if(isYEnd and isZEnd)
            {
                ge3.xstart += isCut[6];
            }
        }
    }}
}
*/
///////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////////////////////
// Pass 3 of the algorithm
///////////////////////////////////////////////////////////////////////////////

__global__
void pass3gpu_blockAccum(
    int nx, int ny, int nz, // which are needed TODO?
    int* triCounter,
    FlyingEdgesAlgorithm::gridEdge* gridEdges,
    int* blockAccum)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (j == 0 && k == 0);
    {
        // TODO get rid of this exp
        blockAccum[0] = 191230;
        blockAccum[1] = 192340;
        blockAccum[2] = 193450;
        blockAccum[3] = 194560;
    }
    return;

    // step 1: accumulate individual y thread
    // step 2: calc block sum
    // step 3: __syncthreads
    // step 4: add to individual y thread


    if(k >= nz)
        return;

    __shared__ int accum[4*FE_BLOCK_WIDTH];

    int tmp;
    int accumX   = 0;
    int accumY   = 0;
    int accumZ   = 0;
    int accumTri = 0;
    for(int j = 0; j != ny; ++j)
    {
        FlyingEdgesAlgorithm::gridEdge& ge = gridEdges[k*ny + j];

        tmp = ge.xstart;
        ge.xstart = accumX;
        accumX += tmp;

        tmp = ge.ystart;
        ge.ystart = accumY;
        accumY += tmp;

        tmp = ge.zstart;
        ge.zstart = accumZ;
        accumZ += tmp;
    }

    if(k < nz-1)
    {
        for(int j = 0; j != ny-1; ++j)
        {
            int& curTriCount = triCounter[k*(ny-1) + j];

            tmp = curTriCount;
            curTriCount = accumTri;
            accumTri += tmp;
        }
    }

    accum[4*threadIdx.z + 0] = accumX;
    accum[4*threadIdx.z + 1] = accumY;
    accum[4*threadIdx.z + 2] = accumZ;
    accum[4*threadIdx.z + 3] = accumTri;

    __syncthreads();

    if(threadIdx.z == 0) // agh!
    {
        for(int idx = 1; idx != blockDim.z; ++idx)
        {
            accum[4*idx + 0] += accum[4*(idx-1) + 0];
            accum[4*idx + 1] += accum[4*(idx-1) + 1];
            accum[4*idx + 2] += accum[4*(idx-1) + 2];
            accum[4*idx + 3] += accum[4*(idx-1) + 3];
        }

        // answer for global accumulation
        blockAccum[4*blockIdx.z + 0] = accum[4*(blockDim.z-1) + 0];
        blockAccum[4*blockIdx.z + 1] = accum[4*(blockDim.z-1) + 1];
        blockAccum[4*blockIdx.z + 2] = accum[4*(blockDim.z-1) + 2];
        blockAccum[4*blockIdx.z + 3] = accum[4*(blockDim.z-1) + 3];
    }

    __syncthreads();

    if(threadIdx.z == 0)
        return;

    bool isEndK = k == nz-1;
    for(int j = 1; j != ny-1; ++j)
    {
        FlyingEdgesAlgorithm::gridEdge& ge = gridEdges[k*ny + j];

        ge.xstart += accum[4*(threadIdx.z-1) + 0];
        ge.ystart += accum[4*(threadIdx.z-1) + 1];
        ge.zstart += accum[4*(threadIdx.z-1) + 2];

        // put z stuff here..
        if(!isEndK)
            triCounter[k*(ny-1) + j] = accum[4*(threadIdx.z-1) + 3];
    }

    FlyingEdgesAlgorithm::gridEdge& ge = gridEdges[k*ny + (ny-1)];
    ge.xstart += accum[4*(threadIdx.z-1) + 0];
    ge.ystart += accum[4*(threadIdx.z-1) + 1];
    ge.zstart += accum[4*(threadIdx.z-1) + 2];
}

__global__ // TODO can split up along j here easy enough.
void pass3gpu_gridAccum(
    int nx, int ny, int nz, // which are needed TODO?
    int* triCounter,
    FlyingEdgesAlgorithm::gridEdge* gridEdges,
    int* blockAccum) // used as input here
{
    // not adding to the first block!
    //
    // add to individual y threads
    int k = (blockIdx.z + 1)*blockDim.z + threadIdx.z;

    if (k >= nz)
        return;

    int addX   = blockAccum[4*blockIdx.z + 0];
    int addY   = blockAccum[4*blockIdx.z + 1];
    int addZ   = blockAccum[4*blockIdx.z + 2];
    int addTri = blockAccum[4*blockIdx.z + 3];

    for(int j = 0; j != ny; ++j)
    {
        FlyingEdgesAlgorithm::gridEdge& ge = gridEdges[k*ny + j];
        ge.xstart += addX;
        ge.ystart += addY;
        ge.zstart += addZ;
    }

    if(k >= nz-1)
        return;

    for(int j = 0; j != ny-1; ++j)
    {
        triCounter[k*(ny-1) + j] += addTri;
    }
}

// Can make prettier?
void FlyingEdgesAlgorithm::pass3()
{
    // Split the z axis
    // Kernel 1: calculate the accum values on block sync
    //           then accum individual values
    // Use that info accum each block (except the first one)
    // Kernel 2: just add values to individual threads
    int tz = FE_BLOCK_WIDTH;

    int numBlocks = (nz + tz - 1) / tz;

    // there are four because: xstart, ystart, zstart, triaccum
    int sizeBlocks = 4 * numBlocks * sizeof(int);

    uint3 gridDim = make_uint3(1, numBlocks, 1); // TODO FIGURE OUT HOW THREADS PER BLOCK
                                                 //      STUFF WORKS with at
                                                 //      3rd dimension..
                                                 //
                                                 //      Blocks can have 3
                                                 //      dimensions
                                                 //
                                                 //      Grids can only have 2
                                                 //      dimensions!
    uint3 blockDim = make_uint3(1, tz, 1);

    int* hostBlockAccum = (int*)malloc(sizeBlocks);

    int* deviceBlockAccum;
    hipMalloc(&deviceBlockAccum, sizeBlocks);

    std::cout << gridDim.x << ", " << gridDim.y << std::endl;
    std::cout << blockDim.x << ", " << blockDim.y << std::endl;

    // Accumulate values locally

    pass3gpu_blockAccum<<<gridDim, blockDim>>>(
        nx, ny, nz,
        triCounter,
        gridEdges,
        deviceBlockAccum);

    hipDeviceSynchronize();

    hipMemcpy(hostBlockAccum, deviceBlockAccum,
               sizeBlocks, hipMemcpyDeviceToHost);


    //////////////////////////////////////////////////////////////
    // WHAT IS GOING ON HERE?
    //////////////////////////////////////////////////////////////

    std::cout  << "SANITY CHECK " << hostBlockAccum[0] << ", " << hostBlockAccum[1] << ", " << hostBlockAccum[2]  << ", " << hostBlockAccum[3] << std::endl;
//    if(err != hipSuccess)
//        std::cout << "AGHHHHHHHHHHHHHHHH" << std::endl;
//    else
//        std::cout << __LINE__ << std::endl;
//    for(int idx = 0; idx != numBlocks; ++idx)
//    {
//        std::cout << hostBlockAccum[4*idx + 3] << std::endl;
//    }

    if(numBlocks != 1)
    {

        // std::partial_sum(2 2 3 4  3  2  2 ) TODO not using it get rid of header
        // goes to         (2 4 7 11 14 16 18)
        // std::partial_sum(hostBlockAccum, hostBlockAccum + numBlocks, hostBlockAccum);

        for(int i = 4; i != 4*numBlocks; i += 4)
        {
            hostBlockAccum[i+0] += hostBlockAccum[i-4];
            hostBlockAccum[i+1] += hostBlockAccum[i-3];
            hostBlockAccum[i+2] += hostBlockAccum[i-2];
            hostBlockAccum[i+3] += hostBlockAccum[i-1];
        }
        // note: the last values in hostBlockAccum should contain total counts

        // The first block is done so it is ignored
        // and the last info in BlockAccum isn't needed (its the total counts)
        hipMemcpy(deviceBlockAccum, hostBlockAccum,
                   sizeBlocks - 4 * sizeof(int), hipMemcpyHostToDevice);

        // Accumulate values from other blocks
        gridDim = make_uint3(1, 1, numBlocks - 1);
        pass3gpu_gridAccum<<<gridDim, blockDim>>>(
            nx, ny, nz,
            triCounter,
            gridEdges,
            deviceBlockAccum);
    }

    // Allocate memory for points, normals and tris
    outputAllocated = true;
    size_t numPoints = hostBlockAccum[4*(numBlocks-1) + 0] +
                       hostBlockAccum[4*(numBlocks-1) + 1] +
                       hostBlockAccum[4*(numBlocks-1) + 2];
    size_t numTris   = hostBlockAccum[4*(numBlocks-1) + 3];

    hipMalloc(&points,  3*sizeof(scalar_t)*numPoints);
    hipMalloc(&normals, 3*sizeof(scalar_t)*numPoints);
    hipMalloc(&tris, 3*sizeof(int)*numTris);

    std::cout << "PASS3 " << numPoints << " " << numTris << std::endl;

    // free memory used in this function
    free(hostBlockAccum);
    hipFree(deviceBlockAccum);

    hipDeviceSynchronize();

}
/*
void FlyingEdgesAlgorithm::pass3()
{
    // Accumulate triangles into triCounter
    size_t tmp;
    size_t triAccum = 0;
    for(size_t k = 0; k != nz-1; ++k) {
    for(size_t j = 0; j != ny-1; ++j)
    {
        size_t& curTriCounter = triCounter[k*(ny-1)+j];

        tmp = curTriCounter;
        curTriCounter = triAccum;
        triAccum += tmp;
    }}

    // accumulate points, filling out starting locations of each gridEdge
    // in the process.
    size_t pointAccum = 0;
    for(size_t k = 0; k != nz; ++k) {
    for(size_t j = 0; j != ny; ++j)
    {
        gridEdge& curGridEdge = gridEdges[k*ny + j];

        tmp = curGridEdge.xstart;
        curGridEdge.xstart = pointAccum;
        pointAccum += tmp;

        tmp = curGridEdge.ystart;
        curGridEdge.ystart = pointAccum;
        pointAccum += tmp;

        tmp = curGridEdge.zstart;
        curGridEdge.zstart = pointAccum;
        pointAccum += tmp;
    }}

    points = std::vector<std::array<scalar_t, 3> >(pointAccum);
    normals = std::vector<std::array<scalar_t, 3> >(pointAccum);
    tris = std::vector<std::array<size_t, 3> >(triAccum);
}
*/
///////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////////////////////
// Pass 4 of the algorithm
///////////////////////////////////////////////////////////////////////////////
void FlyingEdgesAlgorithm::pass4()
{
/* Copy of pass 2, should be similar, just different kernels
 *
    int ty = FE_BLOCK_WIDTH_Y;
    int tz = FE_BLOCK_WIDTH_Z;
    uint3 gridDim = make_uint3(1, ((ny-1) + ty - 1) / ty, ((nz-1) + tz - 1) / tz);
    uint3 blockDim = make_uint3(1, ty, tz);

    pass2gpu_cubeCases<<<gridDim, blockDim>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges,   // modified
        triCounter,  // modified
        cubeCases);  // modified

    // TODO these can be launched and executed independently of each other
    int bw = FE_BLOCK_WIDTH;

    // Making sure that the xz face takes care of the (_, ny-1, nz-1) gridEdge
    // BE CAREFUL. xz takes care of corner. don't use (nz-1)
    pass2gpu_ghost_xz<<<(nz + bw - 1) / bw, bw>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges);
    pass2gpu_ghost_xy<<<((ny-1) + bw - 1) / bw, bw>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges);
*/
}
/*
void FlyingEdgesAlgorithm::pass4()
{
    // For each (j, k):
    //  - For each cube at i, fill out points, normals and triangles owned by
    //    the cube. Each cube is in charge of filling out e0, e3 and e8. Only
    //    in edge cases does it also fill out other edges.
    for(size_t k = 0; k != nz-1; ++k) {
    for(size_t j = 0; j != ny-1; ++j)
    {
        // find adjusted trim values
        size_t xl, xr;
        calcTrimValues(xl, xr, j, k); // xl, xr set in this function

        size_t triIdx = triCounter[k*(ny-1) + j];
        auto curCubeCaseIds = cubeCases.begin() + (nx-1)*(k*(ny-1) + j);

        gridEdge const& ge0 = gridEdges[k*ny + j];
        gridEdge const& ge1 = gridEdges[k*ny + j + 1];
        gridEdge const& ge2 = gridEdges[(k+1)*ny + j];
        gridEdge const& ge3 = gridEdges[(k+1)*ny + j + 1];

        size_t x0counter = 0;
        size_t y0counter = 0;
        size_t z0counter = 0;

        size_t x1counter = 0;
        size_t z1counter = 0;

        size_t x2counter = 0;
        size_t y2counter = 0;

        size_t x3counter = 0;

        bool isYEnd = (j == ny-2);
        bool isZEnd = (k == nz-2);

        for(size_t i = xl; i != xr; ++i)
        {
            bool isXEnd = (i == nx-2);

            uchar caseId = curCubeCaseIds[i];

            if(caseId == 0 || caseId == 255)
            {
                continue;
            }

            const bool* isCut = util::isCut[caseId]; // has 12 elements

            // Most of the information contained in pointCube, isovalCube
            // and gradCube will be used--but not necessarily all. It has
            // not been tested whether or not obtaining only the information
            // needed will provide a significant speedup--but
            // most likely not.
            cube_t        pointCube = image.getPosCube(i, j, k);
            scalarCube_t  isovalCube = image.getValsCube(i, j, k);
            cube_t        gradCube = image.getGradCube(i, j, k);

            // Add Points and normals.
            // Calculate global indices for triangles
            std::array<size_t, 12> globalIdxs;
            if(isCut[0])
            {
                size_t idx = ge0.xstart + x0counter;
                points[idx] = interpolateOnCube(pointCube, isovalCube, 0);
                normals[idx] = interpolateOnCube(gradCube, isovalCube, 0);
                globalIdxs[0] = idx;
                ++x0counter;
            }

            if(isCut[3])
            {
                size_t idx = ge0.ystart + y0counter;
                points[idx] = interpolateOnCube(pointCube, isovalCube, 3);
                normals[idx] = interpolateOnCube(gradCube, isovalCube, 3);
                globalIdxs[3] = idx;
                ++y0counter;
            }

            if(isCut[8])
            {
                size_t idx = ge0.zstart + z0counter;
                points[idx] = interpolateOnCube(pointCube, isovalCube, 8);
                normals[idx] = interpolateOnCube(gradCube, isovalCube, 8);
                globalIdxs[8] = idx;
                ++z0counter;
            }

            // Note:
            //   e1, e5, e9 and e11 will be visited in the next iteration
            //   when they are e3, e7, e8 and 10 respectively. So don't
            //   increment their counters. When the cube is an edge cube,
            //   their counters don't need to be incremented because they
            //   won't be used agin.

            // Manage boundary cases if needed. Otherwise just update
            // globalIdx.
            if(isCut[1])
            {
                size_t idx = ge0.ystart + y0counter;
                if(isXEnd)
                {
                    points[idx] = interpolateOnCube(pointCube, isovalCube, 1);
                    normals[idx] = interpolateOnCube(gradCube, isovalCube, 1);
                    // y0counter counter doesn't need to be incremented
                    // because it won't be used again.
                }
                globalIdxs[1] = idx;
            }

            if(isCut[9])
            {
                size_t idx = ge0.zstart + z0counter;
                if(isXEnd)
                {
                    points[idx] = interpolateOnCube(pointCube, isovalCube, 9);
                    normals[idx] = interpolateOnCube(gradCube, isovalCube, 9);
                    // z0counter doesn't need to in incremented.
                }
                globalIdxs[9] = idx;
            }

            if(isCut[2])
            {
                size_t idx = ge1.xstart + x1counter;
                if(isYEnd)
                {
                    points[idx] = interpolateOnCube(pointCube, isovalCube, 2);
                    normals[idx] = interpolateOnCube(gradCube, isovalCube, 2);
                }
                globalIdxs[2] = idx;
                ++x1counter;
            }

            if(isCut[10])
            {
                size_t idx = ge1.zstart + z1counter;
                if(isYEnd)
                {
                    points[idx] = interpolateOnCube(pointCube, isovalCube, 10);
                    normals[idx] = interpolateOnCube(gradCube, isovalCube, 10);
                }
                globalIdxs[10] = idx;
                ++z1counter;
            }

            if(isCut[4])
            {
                size_t idx = ge2.xstart + x2counter;
                if(isZEnd)
                {
                    points[idx] = interpolateOnCube(pointCube, isovalCube, 4);
                    normals[idx] = interpolateOnCube(gradCube, isovalCube, 4);
                }
                globalIdxs[4] = idx;
                ++x2counter;
            }

            if(isCut[7])
            {
                size_t idx = ge2.ystart + y2counter;
                if(isZEnd)
                {
                    points[idx] = interpolateOnCube(pointCube, isovalCube, 7);
                    normals[idx] = interpolateOnCube(gradCube, isovalCube, 7);
                }
                globalIdxs[7] = idx;
                ++y2counter;
            }

            if(isCut[11])
            {
                size_t idx = ge1.zstart + z1counter;
                if(isXEnd and isYEnd)
                {
                    points[idx] = interpolateOnCube(pointCube, isovalCube, 11);
                    normals[idx] = interpolateOnCube(gradCube, isovalCube, 11);
                    // z1counter does not need to be incremented.
                }
                globalIdxs[11] = idx;
            }

            if(isCut[5])
            {
                size_t idx = ge2.ystart + y2counter;
                if(isXEnd and isZEnd)
                {
                    points[idx] = interpolateOnCube(pointCube, isovalCube, 5);
                    normals[idx] = interpolateOnCube(gradCube, isovalCube, 5);
                    // y2 counter does not need to be incremented.
                }
                globalIdxs[5] = idx;
            }

            if(isCut[6])
            {
                size_t idx = ge3.xstart + x3counter;
                if(isYEnd and isZEnd)
                {
                    points[idx] = interpolateOnCube(pointCube, isovalCube, 6);
                    normals[idx] = interpolateOnCube(gradCube, isovalCube, 6);
                }
                globalIdxs[6] = idx;
                ++x3counter;
            }

            // Add triangles
            const char* caseTri = util::caseTriangles[caseId]; // size 16
            for(int idx = 0; caseTri[idx] != -1; idx += 3)
            {
                tris[triIdx][0] = globalIdxs[caseTri[idx]];
                tris[triIdx][1] = globalIdxs[caseTri[idx+1]];
                tris[triIdx][2] = globalIdxs[caseTri[idx+2]];
                ++triIdx;
            }
        }
    }}
}
///////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////////////////////
// Don't copy points, normals and tris but move the output into a TrianlgeMesh.
///////////////////////////////////////////////////////////////////////////////
util::TriangleMesh FlyingEdgesAlgorithm::moveOutput()
{
    return util::TriangleMesh(std::move(points),
                              std::move(normals),
                              std::move(tris));
}
///////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////////////////////
// Private helper functions
///////////////////////////////////////////////////////////////////////////////

inline uchar
FlyingEdgesAlgorithm::calcCaseEdge(
    bool const& prevEdge,
    bool const& currEdge) const
{
    // o -- is greater than or equal to
    // case 0: (i-1) o-----o (i) | (_,j,k)
    // case 1: (i-1) x-----o (i) | (_,j+1,k)
    // case 2: (i-1) o-----x (i) | (_,j,k+1)
    // case 3: (i-1) x-----x (i) | (_,j+1,k+1)
    if(prevEdge && currEdge)
        return 0;
    if(!prevEdge && currEdge)
        return 1;
    if(prevEdge && !currEdge)
        return 2;
    else // !prevEdge && !currEdge
        return 3;
}

inline uchar
FlyingEdgesAlgorithm::calcCubeCase(
    uchar const& ec0, uchar const& ec1,
    uchar const& ec2, uchar const& ec3) const
{
    // ec0 | (_,j,k)
    // ec1 | (_,j+1,k)
    // ec2 | (_,j,k+1)
    // ec3 | (_,j+1,k+1)

    uchar caseId = 0;
    if((ec0 == 0) || (ec0 == 2)) // 0 | (i,j,k)
        caseId |= 1;
    if((ec0 == 0) || (ec0 == 1)) // 1 | (i+1,j,k)
        caseId |= 2;
    if((ec1 == 0) || (ec1 == 1)) // 2 | (i+1,j+1,k)
        caseId |= 4;
    if((ec1 == 0) || (ec1 == 2)) // 3 | (i,j+1,k)
        caseId |= 8;
    if((ec2 == 0) || (ec2 == 2)) // 4 | (i,j,k+1)
        caseId |= 16;
    if((ec2 == 0) || (ec2 == 1)) // 5 | (i+1,j,k+1)
        caseId |= 32;
    if((ec3 == 0) || (ec3 == 1)) // 6 | (i+1,j+1,k+1)
        caseId |= 64;
    if((ec3 == 0) || (ec3 == 2)) // 7 | (i,j+1,k+1)
        caseId |= 128;
    return caseId;
}

inline void
FlyingEdgesAlgorithm::calcTrimValues(
    size_t& xl, size_t& xr,
    size_t const& j, size_t const& k) const
{
    gridEdge const& ge0 = gridEdges[k*ny + j];
    gridEdge const& ge1 = gridEdges[k*ny + j + 1];
    gridEdge const& ge2 = gridEdges[(k+1)*ny + j];
    gridEdge const& ge3 = gridEdges[(k+1)*ny + j + 1];

    using std::min;
    using std::max;
    xl = min(ge0.xl, min(ge1.xl, min(ge2.xl, ge3.xl)));
    xr = max(ge0.xr, max(ge1.xr, max(ge2.xr, ge3.xr)));
}

inline std::array<scalar_t, 3>
FlyingEdgesAlgorithm::interpolateOnCube(
    cube_t const& pts,
    scalarCube_t const& isovals,
    uchar const& edge) const
{
    uchar i0 = util::edgeVertices[edge][0];
    uchar i1 = util::edgeVertices[edge][1];

    scalar_t weight = (isoval - isovals[i0]) / (isovals[i1] - isovals[i0]);
    return interpolate(pts[i0], pts[i1], weight);
}

inline std::array<scalar_t, 3>
FlyingEdgesAlgorithm::interpolate(
    std::array<scalar_t, 3> const& a,
    std::array<scalar_t, 3> const& b,
    scalar_t const& weight) const
{
    std::array<scalar_t, 3> ret;
    ret[0] = a[0] + (weight * (b[0] - a[0]));
    ret[1] = a[1] + (weight * (b[1] - a[1]));
    ret[2] = a[2] + (weight * (b[2] - a[2]));
    return ret;
}

///////////////////////////////////////////////////////////////////////////////
*/
