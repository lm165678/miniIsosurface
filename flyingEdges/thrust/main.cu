#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <cstdlib>

#include "LoadImage.h"
#include "SaveTriangleMesh.h"

#include "../util/Timer.h"
#include "../mantevoCommon/YAML_Doc.hpp"

#include "config.h"
#include "structs.h"

// TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO
//
// Next: Split the image up and contain ghost sell information.
//
// The problem is that for large input, the entire image will not fit
// into memory.
//
// TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO

int main(int argc, char* argv[])
{
    float isoval;
    bool isovalSet = false;
    char* vtkFile = NULL;
    char* outFile = NULL;
    std::string yamlDirectory = "";
    std::string yamlFileName  = "";
    int process_size = 0; // number of elements to process. If not set,
                          // processes one y x z plane at a time.

    // Read command line arguments
    for(int i=0; i<argc; i++)
    {
        if( (strcmp(argv[i], "-i") == 0) || (strcmp(argv[i], "-input_file") == 0))
        {
            vtkFile = argv[++i];
        }
        else if( (strcmp(argv[i], "-o") == 0) || (strcmp(argv[i], "-output_file") == 0))
        {
            outFile = argv[++i];
        }
        else if( (strcmp(argv[i], "-v") == 0) || (strcmp(argv[i], "-isoval") == 0))
        {
            isovalSet = true;
            isoval = atof(argv[++i]);
        }
        else if( (strcmp(argv[i], "-p") == 0) || (strcmp(argv[i], "-process_size") == 0))
        {
            process_size = atoi(argv[++i]);
        }
        else if( (strcmp(argv[i], "-y") == 0) || (strcmp(argv[i], "-yaml_output_file") == 0))
        {
            std::string wholeFile(argv[++i]);

            std::size_t pos = wholeFile.rfind("/");
            if(pos == std::string::npos)
            {
                yamlDirectory = "./";
                yamlFileName = wholeFile;
            }
            else
            {
                yamlDirectory = wholeFile.substr(0, pos + 1);
                yamlFileName = wholeFile.substr(pos + 1);
            }
        }
        else if( (strcmp(argv[i], "-h") == 0) || (strcmp(argv[i], "-help") == 0))
        {
            std::cout <<
                "Serial Flying Edges Options:"    << std::endl <<
                "  -input_file (-i)"              << std::endl <<
                "  -output_file (-o)"             << std::endl <<
                "  -isoval (-v)"                  << std::endl <<
                "  -process_size (-p)"            << std::endl <<
                "  -yaml_output_file (-y)"        << std::endl <<
                "  -help (-h)"                    << std::endl;
            return 0;
        }
    }

    if(isovalSet == false || vtkFile == NULL || outFile == NULL)
    {
        std::cout << "Error: isoval, input_file and output_file must be set." << std::endl <<
                     "Try -help" << std::endl;
        return 0;
    }

    std::vector<scalar_t> data_host;
    scalar_t spacing_x, spacing_y, spacing_z;
    scalar_t zeropos_x, zeropos_y, zeropos_z;
    int nx, ny, nz;

    util::loadImage_thrust(
        vtkFile,
        data_host,
        spacing_x, spacing_y, spacing_z,
        zeropos_x, zeropos_y, zeropos_z,
        nx, ny, nz);

    YAML_Doc doc("Modified Flying Edges", "0.1", yamlDirectory, yamlFileName);

    doc.add("Modified Flying Edges Algorithm", "cuda, thrust");
    doc.add("Volume image data file path", vtkFile);
    doc.add("Polygonal mesh output file", outFile);
    doc.add("Isoval", isoval);

    if(process_size == 0)
    {
        process_size = ny*nz;
    }
    doc.add("Process Size", process_size);

    doc.add("File x-dimension", nx);
    doc.add("File y-dimension", ny);
    doc.add("File z-dimension", nz);

    ///////////////////////////////////////////////////////////////////////////
    // copy over image data to device and Allocate memory

    // Start timer
    util::Timer run_time;

    util::Timer run_time_allocate_memory;

    vector<scalar_t> image_data(data_host);

    int n = nx*ny*nz;
    int processed = 0;

    // tri_scan
    vector<int> tri_scan(n);
    vector<uchar> cube_ids(n);

    vector<int> a0(n);
    vector<int> b0(n);
    vector<int> c0(n);

    int num_triangles = 0;
    int num_points = 0;

    int max_cur_points = process_size / 100; // used for keeping track of amount
                                             // reserved in pts, nrs vectors
    vector<scalar_t> pts_x(max_cur_points);  // process_size is too big of a guess because
    vector<scalar_t> pts_y(max_cur_points);  // that means every single (x,y,z) is cut.
    vector<scalar_t> pts_z(max_cur_points);  // going with process_size/100 for now.

    vector<scalar_t> nrs_x(max_cur_points);
    vector<scalar_t> nrs_y(max_cur_points);
    vector<scalar_t> nrs_z(max_cur_points);

    int guess_num_pts = n / 100;            // Again, just a guess for amount to reserve.
    host_vector<scalar_t> host_pts_x;     host_pts_x.reserve(guess_num_pts);
    host_vector<scalar_t> host_pts_y;     host_pts_y.reserve(guess_num_pts);
    host_vector<scalar_t> host_pts_z;     host_pts_z.reserve(guess_num_pts);
    host_vector<scalar_t> host_nrs_x;     host_nrs_x.reserve(guess_num_pts);
    host_vector<scalar_t> host_nrs_y;     host_nrs_y.reserve(guess_num_pts);
    host_vector<scalar_t> host_nrs_z;     host_nrs_z.reserve(guess_num_pts);

    run_time_allocate_memory.stop();

    // ----- The algorihtm in a nutshel -----
    // While not all the image has been processed:
    //   First:  fill out pre scan values. That is a0, b0, c0, tri_scan, case_id
    //   Second: scan a0, b0, c0, tri_scan.
    //   Third:  Calculate points and normals. Put to output data at a0, b0 and c0 vals.
    // Free up image data
    // Calculate triangles from a0, b0, c0, tri_scan and case_id
    // --------------------------------------

    util::Timer run_time_points_and_normals;

    std::cout << "num to process: " << n << std::endl;

    while(processed != n)
    {
        int p = std::min(process_size, n - processed);

        // calculate a0, b0, c0, tri_scan

        auto scan_iterator = make_zip_iterator(
            make_tuple(
                a0.begin() + processed,
                b0.begin() + processed,
                c0.begin() + processed,
                tri_scan.begin() + processed,
                cube_ids.begin() + processed));

        // pass1 calculate whether or not a0, b0, c0 is cut, the number of triangles
        // at the (x,y,z) cube and the cube_id.

        transform(
            policy,
            make_counting_iterator(processed),
            make_counting_iterator(processed + p),
            scan_iterator,
            fill_out_pre_scan_values(
                nx, ny, nz,
                isoval,
                image_data.data()));

        ///////////////////////////////////////////////////////////////////////
        // pass 2: scan step + allocate for points and normal values
        // tmp sums
        int tmp_a = a0[processed + p - 1];
        int tmp_b = b0[processed + p - 1];
        int tmp_c = c0[processed + p - 1];
        int tmp_t = tri_scan[processed + p - 1];

        exclusive_scan(
            policy,
            a0.begin() + processed,
            a0.begin() + processed + p,
            a0.begin() + processed,
            num_points);                    // increase starting value

        exclusive_scan(
            policy,
            b0.begin() + processed,
            b0.begin() + processed + p,
            b0.begin() + processed,
            tmp_a + a0[processed + p - 1]); // increase starting value

        exclusive_scan(
            policy,
            c0.begin() + processed,
            c0.begin() + processed + p,
            c0.begin() + processed,
            tmp_b + b0[processed + p - 1]); // increase starting value

        exclusive_scan(
            policy,
            tri_scan.begin() + processed,
            tri_scan.begin() + processed + p,
            tri_scan.begin() + processed,
            num_triangles);

        int prev_num_points = num_points;

        num_triangles = tmp_t + tri_scan[processed + p - 1];
        num_points = tmp_c + c0[processed + p - 1];

        // Allocate points and normals. Triangles will be allocated later
        int cur_num_points = num_points - prev_num_points;
        if(cur_num_points > max_cur_points)
        {
            max_cur_points = cur_num_points;

            pts_x.resize(cur_num_points);
            pts_y.resize(cur_num_points);
            pts_z.resize(cur_num_points);

            nrs_x.resize(cur_num_points);
            nrs_y.resize(cur_num_points);
            nrs_z.resize(cur_num_points);
        }

        ///////////////////////////////////////////////////////////////////////
        // pass 3: calculate points and normal values.
        // Triangles will be done at the end.

        auto pts_nors_info_beg =
            make_zip_iterator(
                make_tuple(
                    make_counting_iterator(processed),
                    scan_iterator));
        auto pts_nors_info_end = pts_nors_info_beg + p;

        for_each(
            policy,
            pts_nors_info_beg,
            pts_nors_info_end,
            calculate_points_and_normals(
                nx, ny, nz,
                spacing_x, spacing_y, spacing_z,
                zeropos_x, zeropos_y, zeropos_z,
                isoval,
                image_data.data(),
                prev_num_points,
                pts_x.data(), pts_y.data(), pts_z.data(),
                nrs_x.data(), nrs_y.data(), nrs_z.data()));

        // TODO make sure host_pts_x has a good initial guess to the size.
        // Its on the host, so can make big lar
        host_pts_x.resize(num_points);
        host_pts_y.resize(num_points);
        host_pts_z.resize(num_points);
        host_nrs_x.resize(num_points);
        host_nrs_y.resize(num_points);
        host_nrs_z.resize(num_points);

        int& cnp = cur_num_points; // just making the names shorter
        int& pnp = prev_num_points;

        thrust::copy(pts_x.begin(), pts_x.begin() + cnp, host_pts_x.begin() + pnp);
        thrust::copy(pts_y.begin(), pts_y.begin() + cnp, host_pts_y.begin() + pnp);
        thrust::copy(pts_z.begin(), pts_z.begin() + cnp, host_pts_z.begin() + pnp);
        thrust::copy(nrs_x.begin(), nrs_x.begin() + cnp, host_nrs_x.begin() + pnp);
        thrust::copy(nrs_y.begin(), nrs_y.begin() + cnp, host_nrs_y.begin() + pnp);
        thrust::copy(nrs_z.begin(), nrs_z.begin() + cnp, host_nrs_z.begin() + pnp);

        ///////////////////////////////////////////////////////////////////////
        processed += p;
        std::cout << "num processed: " << processed << std::endl;
    }

    run_time_points_and_normals.stop();

    // Free up memory
    image_data.resize(0);

    pts_x.resize(0);   nrs_x.resize(0);
    pts_y.resize(0);   nrs_y.resize(0);
    pts_z.resize(0);   nrs_z.resize(0);

    ///////////////////////////////////////////////////////////////////////////
    // Set and allocate triangles
    util::Timer run_time_set_triangles;

    vector<int> trs0(num_triangles);
    vector<int> trs1(num_triangles);
    vector<int> trs2(num_triangles);

    for_each(
        policy,
        make_counting_iterator(0),
        make_counting_iterator((nx-1)*(ny-1)*(nz-1)),
        set_triangles(
            trs0.data(), trs1.data(), trs2.data(),
            nx, ny, nz,
            a0.data(), b0.data(), c0.data(),
            cube_ids.data(),
            tri_scan.data()));

    host_vector<int> host_trs_0(trs0);
    host_vector<int> host_trs_1(trs1);
    host_vector<int> host_trs_2(trs2);

    run_time_set_triangles.stop();
    run_time.stop();

    doc.add("Number of vertices in mesh", num_points);
    doc.add("Number of triangles in mesh", num_triangles);

    doc.add("Allocate Memory", "");
    doc.get("Allocate Memory")->add(
        "CPU Time (clicks)", run_time_allocate_memory.getTotalTicks());
    doc.get("Allocate Memory")->add(
        "CPU Time (seconds)", run_time_allocate_memory.getCPUtime());
    doc.get("Allocate Memory")->add(
        "Wall Time (seconds)", run_time_allocate_memory.getWallTime());

    doc.add("Set Points and Normals", "");
    doc.get("Set Points and Normals")->add(
        "CPU Time (clicks)", run_time_points_and_normals.getTotalTicks());
    doc.get("Set Points and Normals")->add(
        "CPU Time (seconds)", run_time_points_and_normals.getCPUtime());
    doc.get("Set Points and Normals")->add(
        "Wall Time (seconds)", run_time_points_and_normals.getWallTime());

    doc.add("Set Triangles", "");
    doc.get("Set Triangles")->add(
        "CPU Time (clicks)", run_time_set_triangles.getTotalTicks());
    doc.get("Set Triangles")->add(
        "CPU Time (seconds)", run_time_set_triangles.getCPUtime());
    doc.get("Set Triangles")->add(
        "Wall Time (seconds)", run_time_set_triangles.getWallTime());

    doc.add("Total Program CPU Time (clicks)", run_time.getTotalTicks());
    doc.add("Total Program CPU Time (seconds)", run_time.getCPUtime());
    doc.add("Total Program WALL Time (seconds)", run_time.getWallTime());

    std::cout << doc.generateYAML();

    util::saveTriangleMesh(
        outFile,
        host_pts_x, host_pts_y, host_pts_z,
        host_nrs_x, host_nrs_y, host_nrs_z,
        host_trs_0, host_trs_1, host_trs_2);

/*
 ******* Metrics to test correctness*******
    scalar_t sum_pts = 0.0;
    scalar_t sum_nrs = 0.0;
    for(int w = 0; w != num_points; ++w)
    {
        sum_pts += host_pts_x[w] + host_pts_y[w] + host_pts_z[w];
        sum_nrs += host_nrs_x[w] + host_nrs_y[w] + host_nrs_z[w];

        while(sum_pts > 500000000)
            sum_pts -=  500000000;
        while(sum_pts <-500000000)
            sum_pts +=  500000000;
        while(sum_nrs > 500000000)
            sum_nrs -=  500000000;
        while(sum_nrs <-500000000)
            sum_nrs +=  500000000;
    }

    scalar_t sum_tri_pts = 0.0;
    for(int v = 0; v != num_triangles; ++v)
    {
        int p1 = host_trs0[v];
        int p2 = host_trs1[v];
        int p3 = host_trs2[v];

        sum_tri_pts += host_pts_x[p1] + host_pts_y[p1] + host_pts_z[p1] +
                       host_pts_x[p2] + host_pts_y[p2] + host_pts_z[p2] +
                       host_pts_x[p3] + host_pts_y[p3] + host_pts_z[p3];


        while(sum_tri_pts > 500000000)
            sum_tri_pts -=  500000000;
        while(sum_tri_pts <-500000000)
            sum_tri_pts +=  500000000;
    }

    std::cout << "num points " << num_points << ", " << host_pts_x.size() << std::endl;
    std::cout << "num triangles " << num_triangles << ", " << host_trs0.size() << std::endl;
    std::cout << "sum points " << sum_pts << std::endl;
    std::cout << "sum normals " << sum_nrs << std::endl;
    std::cout << "sum triangle points " << sum_tri_pts << std::endl;
*/
}


