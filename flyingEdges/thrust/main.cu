#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <cstdlib>

#include "LoadImage.h"
#include "SaveTriangleMesh.h"

#include "../util/Timer.h"
#include "../mantevoCommon/YAML_Doc.hpp"

#include "config.h"
#include "structs.h"

// TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO
//
// While correct, this code is slower than the serial version.
// The most likely culprit is that way too much memory is being
// allocated upfront.
//
// Instead of setting points and normals in the first part--and
// thereby allocating way too much memory that is mostly filled with
// unused values, the next version should only fill out a0, b0, c0,
// triscan and caseid on the first part. Then scan them. After the
// scan step, allocate however much memory available and set points
// and normals and then triangles.
//
// The limiting factor is not how much computation can be done but
// memory.
//
// TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO TODO

int main(int argc, char* argv[])
{
    float isoval;
    bool isovalSet = false;
    char* vtkFile = NULL;
    char* outFile = NULL;
    std::string yamlDirectory = "";
    std::string yamlFileName  = "";
    int process_size = 0; // number of elements to process. If not set,
                          // processes one y x z plane at a time.

    // Read command line arguments
    for(int i=0; i<argc; i++)
    {
        if( (strcmp(argv[i], "-i") == 0) || (strcmp(argv[i], "-input_file") == 0))
        {
            vtkFile = argv[++i];
        }
        else if( (strcmp(argv[i], "-o") == 0) || (strcmp(argv[i], "-output_file") == 0))
        {
            outFile = argv[++i];
        }
        else if( (strcmp(argv[i], "-v") == 0) || (strcmp(argv[i], "-isoval") == 0))
        {
            isovalSet = true;
            isoval = atof(argv[++i]);
        }
        else if( (strcmp(argv[i], "-p") == 0) || (strcmp(argv[i], "-process_size") == 0))
        {
            process_size = atoi(argv[++i]);
        }
        else if( (strcmp(argv[i], "-y") == 0) || (strcmp(argv[i], "-yaml_output_file") == 0))
        {
            std::string wholeFile(argv[++i]);

            std::size_t pos = wholeFile.rfind("/");
            if(pos == std::string::npos)
            {
                yamlDirectory = "./";
                yamlFileName = wholeFile;
            }
            else
            {
                yamlDirectory = wholeFile.substr(0, pos + 1);
                yamlFileName = wholeFile.substr(pos + 1);
            }
        }
        else if( (strcmp(argv[i], "-h") == 0) || (strcmp(argv[i], "-help") == 0))
        {
            std::cout <<
                "Serial Flying Edges Options:"    << std::endl <<
                "  -input_file (-i)"              << std::endl <<
                "  -output_file (-o)"             << std::endl <<
                "  -isoval (-v)"                  << std::endl <<
                "  -process_size (-p)"            << std::endl <<
                "  -yaml_output_file (-y)"        << std::endl <<
                "  -help (-h)"                    << std::endl;
            return 0;
        }
    }

    if(isovalSet == false || vtkFile == NULL || outFile == NULL)
    {
        std::cout << "Error: isoval, input_file and output_file must be set." << std::endl <<
                     "Try -help" << std::endl;
        return 0;
    }

    std::vector<scalar_t> data_host;
    scalar_t spacing_x, spacing_y, spacing_z;
    scalar_t zeropos_x, zeropos_y, zeropos_z;
    int nx, ny, nz;

    util::loadImage_thrust(
        vtkFile,
        data_host,
        spacing_x, spacing_y, spacing_z,
        zeropos_x, zeropos_y, zeropos_z,
        nx, ny, nz);

    YAML_Doc doc("Modified Flying Edges", "0.1", yamlDirectory, yamlFileName);

    doc.add("Modified Flying Edges Algorithm", "cuda, thrust");
    doc.add("Volume image data file path", vtkFile);
    doc.add("Polygonal mesh output file", outFile);
    doc.add("Isoval", isoval);

    if(process_size == 0)
    {
        process_size = ny*nz;
    }
    doc.add("Process Size", process_size);

    doc.add("File x-dimension", nx);
    doc.add("File y-dimension", ny);
    doc.add("File z-dimension", nz);

    ///////////////////////////////////////////////////////////////////////////
    // copy over image data to device and Allocate memory

    // Start timer
    util::Timer run_time;

    util::Timer run_time_allocate_memory;

    vector<scalar_t> image_data(data_host);

    int n = nx*ny*nz;
    int processed = 0;

    // a_xyz
    vector<scalar_t> ax(process_size);  // points    ; will be set to -1 if not needed
    vector<scalar_t> ay(process_size);  //           ; the others don't matter
    vector<scalar_t> az(process_size);
    vector<scalar_t> axn(process_size); // normals
    vector<scalar_t> ayn(process_size);
    vector<scalar_t> azn(process_size);

    // b_xyz
    vector<scalar_t> bx(process_size); // has to be set to -1 if not used
    vector<scalar_t> by(process_size);
    vector<scalar_t> bz(process_size);
    vector<scalar_t> bxn(process_size);
    vector<scalar_t> byn(process_size);
    vector<scalar_t> bzn(process_size);

    // c_xyz
    vector<scalar_t> cx(process_size); // has to be set to -1 if not used
    vector<scalar_t> cy(process_size);
    vector<scalar_t> cz(process_size);
    vector<scalar_t> cxn(process_size);
    vector<scalar_t> cyn(process_size);
    vector<scalar_t> czn(process_size);

    // cube_ids
    vector<uchar> cube_ids(n);

    // tri_scan
    vector<int> tri_scan(n); // needs to be int for scan step

    vector<int> a0(n);
    vector<int> b0(n);
    vector<int> c0(n);

    int num_triangles = 0;
    int num_points = 0;

    scalar_t default_value = std::min(
        zeropos_x,
        std::min(zeropos_y, zeropos_z)) - 1;

    int max_cur_points = process_size;
    vector<scalar_t> pts_x(process_size); // guess at the size, may make it bigger
    vector<scalar_t> pts_y(process_size); // during algorithm. But probably already
    vector<scalar_t> pts_z(process_size); // too big of a guess

    vector<scalar_t> nrs_x(process_size);
    vector<scalar_t> nrs_y(process_size);
    vector<scalar_t> nrs_z(process_size);

    host_vector<scalar_t> host_pts_x(0);
    host_vector<scalar_t> host_pts_y(0);
    host_vector<scalar_t> host_pts_z(0);
    host_vector<scalar_t> host_nrs_x(0);
    host_vector<scalar_t> host_nrs_y(0);
    host_vector<scalar_t> host_nrs_z(0);

    run_time_allocate_memory.stop();

    util::Timer run_time_points_and_normals;

    while(processed != n)
    {
        int p = std::min(process_size, n - processed);

        auto a_iter = make_zip_iterator(
            make_tuple(
                ax.begin(),  ay.begin(),  az.begin(),
                axn.begin(), ayn.begin(), azn.begin()));
        auto b_iter = make_zip_iterator(
            make_tuple(
                bx.begin(),  by.begin(),  bz.begin(),
                bxn.begin(), byn.begin(), bzn.begin()));
        auto c_iter = make_zip_iterator(
            make_tuple(
                cx.begin(),  cy.begin(),  cz.begin(),
                cxn.begin(), cyn.begin(), czn.begin()));

        auto pts_nors_plus = make_zip_iterator(
            make_tuple(
                a_iter,
                b_iter,
                c_iter,
                cube_ids.begin() + processed,
                tri_scan.begin() + processed));

        transform(
            policy,
            make_counting_iterator(processed),     // Will calculate v0, ..., v7 from
            make_counting_iterator(processed + p), // function. As well as gradient vs
            pts_nors_plus,
            abc_transform(
                nx, ny, nz,
                spacing_x, spacing_y, spacing_z,
                zeropos_x, zeropos_y, zeropos_z,
                isoval, image_data.data()));

        ///////////////////////////////////////////////////////////////////////////

        // tmp sums
        int np_a_temp = (ax[p-1] != -1);
        int np_b_temp = (bx[p-1] != -1);
        int np_c_temp = (cx[p-1] != -1);
        int num_triangles_temp = tri_scan[processed + p - 1];

        auto ax_mod_iter = make_transform_iterator(
            ax.begin(),
            neq(default_value));
        auto bx_mod_iter = make_transform_iterator(
            bx.begin(),
            neq(default_value));
        auto cx_mod_iter = make_transform_iterator(
            cx.begin(),
            neq(default_value));

        exclusive_scan(
            policy,
            ax_mod_iter,
            ax_mod_iter + p,
            a0.begin() + processed,
            num_points);

        exclusive_scan(
            policy,
            bx_mod_iter,
            bx_mod_iter + p,
            b0.begin() + processed,
            np_a_temp + a0[processed + p - 1]); // increase starting value

        exclusive_scan(
            policy,
            cx_mod_iter,
            cx_mod_iter + p,
            c0.begin() + processed,
            np_b_temp + b0[processed + p - 1]); // increase starting value

        // Don't need num triangles; can do based off of cubeIds!
        exclusive_scan(
            policy,
            tri_scan.begin() + processed,
            tri_scan.begin() + processed + p,
            tri_scan.begin() + processed,
            num_triangles);

        int prev_num_points = num_points;
        // final values of sums
        num_triangles = num_triangles_temp + tri_scan[processed + p - 1];
        num_points = np_c_temp + c0[processed + p - 1];

        ///////////////////////////////////////////////////////////////////////////
        // Allocate points and normals. Triangles will be allocated later
        int cur_num_points = num_points - prev_num_points;

        if(cur_num_points > max_cur_points)
        {
            max_cur_points = cur_num_points;

            pts_x.resize(cur_num_points);
            pts_y.resize(cur_num_points);
            pts_z.resize(cur_num_points);

            nrs_x.resize(cur_num_points);
            nrs_y.resize(cur_num_points);
            nrs_z.resize(cur_num_points);
        }

        ///////////////////////////////////////////////////////////////////////////
        // Set points and normals
        auto indexer = make_zip_iterator(
            make_tuple(
                a0.begin() + processed,
                b0.begin() + processed,
                c0.begin() + processed));

        auto indexer_plus_info_iterator = make_zip_iterator(
            make_tuple(
                indexer,
                a_iter,
                b_iter,
                c_iter));

        for_each(
            policy,
            indexer_plus_info_iterator,
            indexer_plus_info_iterator + p,
            set_points_and_normals(
                pts_x.data(), pts_y.data(), pts_z.data(),
                nrs_x.data(), nrs_y.data(), nrs_z.data(),
                default_value,
                prev_num_points));

        host_pts_x.resize(num_points);
        host_pts_y.resize(num_points);
        host_pts_z.resize(num_points);
        host_nrs_x.resize(num_points);
        host_nrs_y.resize(num_points);
        host_nrs_z.resize(num_points);

        int& cnp = cur_num_points; // just making the names shorter
        int& pnp = prev_num_points;

        thrust::copy(pts_x.begin(), pts_x.begin() + cnp, host_pts_x.begin() + pnp);
        thrust::copy(pts_y.begin(), pts_y.begin() + cnp, host_pts_y.begin() + pnp);
        thrust::copy(pts_z.begin(), pts_z.begin() + cnp, host_pts_z.begin() + pnp);
        thrust::copy(nrs_x.begin(), nrs_x.begin() + cnp, host_nrs_x.begin() + pnp);
        thrust::copy(nrs_y.begin(), nrs_y.begin() + cnp, host_nrs_y.begin() + pnp);
        thrust::copy(nrs_z.begin(), nrs_z.begin() + cnp, host_nrs_z.begin() + pnp);

        processed += p;

        std::cout << "num processed: " << processed << std::endl;
    }

    run_time_points_and_normals.stop();

    // Free up memory
    image_data.resize(0);

    pts_x.resize(0);   nrs_x.resize(0);
    pts_y.resize(0);   nrs_y.resize(0);
    pts_z.resize(0);   nrs_z.resize(0);

    ax.resize(0);    bx.resize(0);    cx.resize(0);
    ay.resize(0);    by.resize(0);    cy.resize(0);
    az.resize(0);    bz.resize(0);    cz.resize(0);
    axn.resize(0);   bxn.resize(0);   cxn.resize(0);
    ayn.resize(0);   byn.resize(0);   cyn.resize(0);
    azn.resize(0);   bzn.resize(0);   czn.resize(0);

    ///////////////////////////////////////////////////////////////////////////
    // Set and allocate triangles

    util::Timer run_time_triangles;

    vector<int> trs0(num_triangles);
    vector<int> trs1(num_triangles);
    vector<int> trs2(num_triangles);

    for_each(
        policy,
        make_counting_iterator(0),
        make_counting_iterator((nx-1)*(ny-1)*(nz-1)),
        set_triangles(
            trs0.data(), trs1.data(), trs2.data(),
            nx, ny, nz,
            a0.data(), b0.data(), c0.data(),
            cube_ids.data(),
            tri_scan.data()));

    host_vector<int> host_trs_0(trs0);
    host_vector<int> host_trs_1(trs1);
    host_vector<int> host_trs_2(trs2);

    run_time_triangles.stop();
    run_time.stop();

    doc.add("Number of vertices in mesh", num_points);
    doc.add("Number of triangles in mesh", num_triangles);

    doc.add("Allocate Memory", "");
    doc.get("Allocate Memory")->add(
        "CPU Time (clicks)", run_time_allocate_memory.getTotalTicks());
    doc.get("Allocate Memory")->add(
        "CPU Time (seconds)", run_time_allocate_memory.getCPUtime());
    doc.get("Allocate Memory")->add(
        "Wall Time (seconds)", run_time_allocate_memory.getWallTime());

    doc.add("Set Points and Normals", "");
    doc.get("Set Points and Normals")->add(
        "CPU Time (clicks)", run_time_points_and_normals.getTotalTicks());
    doc.get("Set Points and Normals")->add(
        "CPU Time (seconds)", run_time_points_and_normals.getCPUtime());
    doc.get("Set Points and Normals")->add(
        "Wall Time (seconds)", run_time_points_and_normals.getWallTime());

    doc.add("Set Triangles", "");
    doc.get("Set Triangles")->add(
        "CPU Time (clicks)", run_time_triangles.getTotalTicks());
    doc.get("Set Triangles")->add(
        "CPU Time (seconds)", run_time_triangles.getCPUtime());
    doc.get("Set Triangles")->add(
        "Wall Time (seconds)", run_time_triangles.getWallTime());

    doc.add("Total Program CPU Time (clicks)", run_time.getTotalTicks());
    doc.add("Total Program CPU Time (seconds)", run_time.getCPUtime());
    doc.add("Total Program WALL Time (seconds)", run_time.getWallTime());

    std::cout << doc.generateYAML();

    util::saveTriangleMesh(
        outFile,
        host_pts_x, host_pts_y, host_pts_z,
        host_nrs_x, host_nrs_y, host_nrs_z,
        host_trs_0, host_trs_1, host_trs_2);

/*
 ******* Metrics to test correctness*******
    scalar_t sum_pts = 0.0;
    scalar_t sum_nrs = 0.0;
    for(int w = 0; w != num_points; ++w)
    {
        sum_pts += host_pts_x[w] + host_pts_y[w] + host_pts_z[w];
        sum_nrs += host_nrs_x[w] + host_nrs_y[w] + host_nrs_z[w];

        while(sum_pts > 500000000)
            sum_pts -=  500000000;
        while(sum_pts <-500000000)
            sum_pts +=  500000000;
        while(sum_nrs > 500000000)
            sum_nrs -=  500000000;
        while(sum_nrs <-500000000)
            sum_nrs +=  500000000;
    }

    scalar_t sum_tri_pts = 0.0;
    for(int v = 0; v != num_triangles; ++v)
    {
        int p1 = host_trs0[v];
        int p2 = host_trs1[v];
        int p3 = host_trs2[v];

        sum_tri_pts += host_pts_x[p1] + host_pts_y[p1] + host_pts_z[p1] +
                       host_pts_x[p2] + host_pts_y[p2] + host_pts_z[p2] +
                       host_pts_x[p3] + host_pts_y[p3] + host_pts_z[p3];


        while(sum_tri_pts > 500000000)
            sum_tri_pts -=  500000000;
        while(sum_tri_pts <-500000000)
            sum_tri_pts +=  500000000;
    }

    std::cout << "num points " << num_points << ", " << host_pts_x.size() << std::endl;
    std::cout << "num triangles " << num_triangles << ", " << host_trs0.size() << std::endl;
    std::cout << "sum points " << sum_pts << std::endl;
    std::cout << "sum normals " << sum_nrs << std::endl;
    std::cout << "sum triangle points " << sum_tri_pts << std::endl;
*/
}


