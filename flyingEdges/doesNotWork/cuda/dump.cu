
#include <hip/hip_runtime.h>
///////////////////////////////////////////////////////////////////////////////
// *Time: 5e-5 seconds
///////////////////////////////////////////////////////////////////////////////
//

/*
__global__
void pass1gpu(
    scalar_t* pointValues,  // input
    int nx, int ny, int nz, // input
    scalar_t isoval,        // input
    uchar* edgeCases)       // output
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if(j >= ny || k >= nz)
        return;

    edgeCases[k*ny*(nx-1) + j*(nx-1) + i] =
        calcCaseEdge(
            pointValues[k*ny*nx + j*nx + i + 0] >= isoval,
            pointValues[k*ny*nx + j*nx + i + 1] >= isoval);
}
*/

/*
__global__
void pass1gpu(
    scalar_t* pointValues,  // input
    scalar_t isoval,        // input
    uchar* edgeCases)       // output
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    int k = blockIdx.z;

    int ny = blockDim.y;
    int nx = blockDim.x + 1;

    edgeCases[k*ny*(nx-1) + j*(nx-1) + i] =
        calcCaseEdge(
            pointValues[k*ny*nx + j*nx + i + 0] >= isoval,
            pointValues[k*ny*nx + j*nx + i + 1] >= isoval);
}

void FlyingEdgesAlgorithm::pass1()
{
    dim3 dims = make_uint3(nx-1, ny, nz);
    pass1gpu<<<dims, 1>>>(pointValues, isoval, edgeCases);
}
*/
///////////////////////////////////////////////////////////////////////////////
// *Time: 8.2e-5
///////////////////////////////////////////////////////////////////////////////

/*
__global__
void pass1gpu1(
    scalar_t* pointValues,  // input
    scalar_t isoval,        // input
    uchar* edgeCases)       // output
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    int k = blockIdx.z;

    int ny = blockDim.y;
    int nx = blockDim.x + 1;

    edgeCases[k*ny*(nx-1) + j*(nx-1) + i] =
        calcCaseEdge(
            pointValues[k*ny*nx + j*nx + i + 0] >= isoval,
            pointValues[k*ny*nx + j*nx + i + 1] >= isoval);
}

__global__
void pass1gpu2(
    uchar* edgeCases,    // input
    int nx,              // input
    FlyingEdgesAlgorithm::gridEdge* gridEdges) // output
{
    int j = blockIdx.y;
    int k = blockIdx.z;

    int ny = blockDim.y;

    FlyingEdgesAlgorithm::gridEdge& grid = gridEdges[k*ny + j];

    for(int i = 0; i != nx-1; ++i)
    {
        uchar const& edge = edgeCases[k*ny*(nx-1) + j*(nx-1) + i];
        if(edge == 1 || edge == 2)
        {
            grid.xl = i;
            break;
        }
    }

    for(int i = nx-2; i != -1; ++i)
    {
        uchar const& edge = edgeCases[k*ny*(nx-1) + j*(nx-1) + i];
        if(edge == 1 || edge == 2)
        {
            grid.xr = i;
            break;
        }
    }
}

__global__
void pass1gpu222(
    scalar_t* pointValues,                     // input
    scalar_t isoval,                           // input
    int nx,                                    // input
    int ny,
    uchar* edgeCases,                          // output
    FlyingEdgesAlgorithm::gridEdge* gridEdges) // output
{
//    int j = blockIdx.y;
//    int k = blockIdx.z;
//
//    int ny = blockDim.y;
    int j = threadIdx.y;
    int k = threadIdx.z;

    scalar_t* curPointValues = pointValues + k*nx*ny + j*nx;
    uchar* curEdgeCases = edgeCases + k*(nx-1)*ny + j*(nx-1);
    FlyingEdgesAlgorithm::gridEdge& curGridEdge = gridEdges[k*ny + j];

    bool isGE[2];
    isGE[0] = (curPointValues[0] >= isoval);
    for(int i = 1; i != nx; ++i)
    {
        isGE[i%2] = (curPointValues[i] >= isoval);
        curEdgeCases[i-1] = calcCaseEdge(isGE[(i+1)%2], isGE[i%2]);

        if(curEdgeCases[i-1] == 1 || curEdgeCases[i-1] == 2)
        {
            if(curGridEdge.xl == 0)
                curGridEdge.xl = i-1;
            curGridEdge.xr = i;
        }
    }
}

__global__
void pass1gpu333(
    scalar_t* pointValues,                     // input
    scalar_t isoval,                           // input
    int nx, int ny, int nz,                    // input
    uchar* edgeCases,                          // output
    FlyingEdgesAlgorithm::gridEdge* gridEdges) // output
{
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (j >= ny || k >= nz)
        return;

    scalar_t* curPointValues = pointValues + k*nx*ny + j*nx;
    uchar* curEdgeCases = edgeCases + k*(nx-1)*ny + j*(nx-1);
    FlyingEdgesAlgorithm::gridEdge& curGridEdge = gridEdges[k*ny + j];

    bool isGE[2];
    isGE[0] = (curPointValues[0] >= isoval);
//    for(int i = 1; i != nx; ++i)
//    {
//        isGE[i%2] = (curPointValues[i] >= isoval);
//        curEdgeCases[i-1] = calcCaseEdge(isGE[(i+1)%2], isGE[i%2]);
//
//        if(curEdgeCases[i-1] == 1 || curEdgeCases[i-1] == 2)
//        {
//            if(curGridEdge.xl == 0)
//                curGridEdge.xl = i-1;
//            curGridEdge.xr = i;
//        }
//    }
}
*/
