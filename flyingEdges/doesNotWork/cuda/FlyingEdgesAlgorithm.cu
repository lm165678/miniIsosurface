#include "hip/hip_runtime.h"
/*
 * flyingEdgesAlgorithm.cpp
 *
 *  Created on: Feb 17, 2017
 *      Author: dbourge
 */
#include "FlyingEdgesAlgorithm.h"

#include "CudaMarchingCubesTables.h"

#include <numeric>

#include <algorithm> // TODO
#include <iostream> // TODO

#define MAX_X_GRID 65535 // says larger but doesnt work if larger..
#define MAX_Y_GRID 65535
#define MAX_Z_GRID 65535
#define MAX_X_BLOCK 1024
#define MAX_Y_BLOCK 1024
#define MAX_Z_BLOCK 64
#define MAX_THREAD_PER_BLOCK 1024

#define DEBUG true

// TODO figure out how to handle errors

bool validKernelSize(uint3 const& gridDim, uint3 const& blockDim)
{
    if(gridDim.x > MAX_X_GRID)
        return false;
    if(gridDim.y > MAX_Y_GRID)
        return false;
    if(gridDim.z > MAX_Z_GRID)
        return false;

    if(blockDim.x > MAX_X_BLOCK)
        return false;
    if(blockDim.y > MAX_Y_BLOCK)
        return false;
    if(blockDim.z > MAX_Z_BLOCK)
        return false;

    if(blockDim.x * blockDim.y * blockDim.z > MAX_THREAD_PER_BLOCK)
        return false;

    return true;
}

// TODO make sure pointValues stored in const memory

///////////////////////////////////////////////////////////////////////////////
// Pass 1 of the algorithm
///////////////////////////////////////////////////////////////////////////////

__device__
uchar calcCaseEdge(
    bool const& prevEdge,
    bool const& currEdge)
{
    // o -- is greater than or equal to
    // case 0: (i-1) o-----o (i) | (_,j,k)
    // case 1: (i-1) x-----o (i) | (_,j+1,k)
    // case 2: (i-1) o-----x (i) | (_,j,k+1)
    // case 3: (i-1) x-----x (i) | (_,j+1,k+1)
    if(prevEdge && currEdge)
        return 0;
    if(!prevEdge && currEdge)
        return 1;
    if(prevEdge && !currEdge)
        return 2;
    else // !prevEdge && !currEdge
        return 3;
}

__global__
void pass1gpu_edgeCases(
    scalar_t* pointValues,
    scalar_t isoval,
    int nx, int ny,
    uchar* edgeCases)
{
    // Each row has several blocks
    // Each thread is one point

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y;
    int k = blockIdx.z;

    __shared__ bool isGE[FE_BLOCK_WIDTH_PLUS_ONE];

    if(i < nx)
        isGE[threadIdx.x] = pointValues[k*nx*ny + j*nx + i] >= isoval;

    if(threadIdx.x == 0 && i < nx-1)
    {
        isGE[blockDim.x] = pointValues[k*nx*ny + j*nx + i + blockDim.x] >= isoval;
    }

    __syncthreads();

    if(i < nx-1)
    {
        uchar caseEdge = calcCaseEdge(isGE[threadIdx.x], isGE[threadIdx.x + 1]);
        edgeCases[k*(nx-1)*ny + j*(nx-1) + i] = caseEdge;
    }
}

__global__
void pass1gpu_trim(
    int nx, int ny, int nz,                    // input
    uchar* edgeCases,                          // input
    FlyingEdgesAlgorithm::gridEdge* gridEdges) // output
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if(j >= ny || k >= nz)
        return;

    size_t xl = nx;
    size_t xr = 0;

    uchar* curEdgeCases = edgeCases + k*(nx-1)*ny + j*(nx-1);

    for(int i = 0; i != nx-1; ++i)
    {
        if(curEdgeCases[i] == 1 || curEdgeCases[i] == 2)
        {
            if(xl == nx)
                xl = i;
            xr = i+1;
        }
    }

    gridEdges[k*ny + j].xl = xl;
    gridEdges[k*ny + j].xr = xr;
}

void FlyingEdgesAlgorithm::pass1()
{
    int tx = FE_BLOCK_WIDTH;
    uint3 gridDim = make_uint3(((nx-1) + tx - 1) / tx, ny, nz);
    uint3 blockDim = make_uint3(tx, 1, 1);

    if(!validKernelSize(gridDim, blockDim))
        std::cout << "GAHHHHHHHHHHHH GP2 ENGINE " << __LINE__ << std::endl; // TODO

    pass1gpu_edgeCases<<<gridDim, blockDim>>>(
        pointValues,
        isoval,
        nx, ny,
        edgeCases);

    int ty = FE_BLOCK_WIDTH_Y;
    int tz = FE_BLOCK_WIDTH_Z;
    gridDim = make_uint3((ny + ty - 1) / ty, (nz + tz - 1) / tz, 1);
    blockDim = make_uint3(ty, tz, 1);

    if(!validKernelSize(gridDim, blockDim))
        std::cout << "GAHHHHHHHHHHHH GP2 ENGINE " << __LINE__ << std::endl; // TODO

    pass1gpu_trim<<<gridDim, blockDim>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges);

    hipDeviceSynchronize();

    if(DEBUG)
    {
        int numGE = nz*ny;
        gridEdge* hostGEs = (gridEdge*)malloc(numGE*sizeof(gridEdge));
        hipMemcpy(hostGEs, gridEdges, numGE*sizeof(gridEdge),
                   hipMemcpyDeviceToHost);

        int numCubes=(nx-1)*ny*nz;
        size_t count = 0;
        uchar* hoseEdgeCases = (uchar*)malloc(numCubes*sizeof(uchar));
        hipMemcpy(hoseEdgeCases, edgeCases, numCubes*sizeof(uchar),
                   hipMemcpyDeviceToHost);
        for(int idx = 0; idx != numCubes; ++idx)
        {
            uchar const& val = hoseEdgeCases[idx];
            count += val;
        }
        std::cout << "Edgecase counter: " << count << std::endl;
        free(hoseEdgeCases);


        size_t countL = 0;
        size_t countR = 0;
        for(int idx = 0; idx != numGE; ++idx)
        {
            countL += hostGEs[idx].xl;
            countR += hostGEs[idx].xr;
        }

        std::cout << "xl, xr: " << countL << ", " << countR << std::endl;

        free(hostGEs);
    }
}

///////////////////////////////////////////////////////////////////////////////
// Pass 2 of the algorithm
///////////////////////////////////////////////////////////////////////////////

__device__
void calcTrimValues(
    int& xl, int& xr,
    FlyingEdgesAlgorithm::gridEdge const& ge0,
    FlyingEdgesAlgorithm::gridEdge const& ge1,
    FlyingEdgesAlgorithm::gridEdge const& ge2,
    FlyingEdgesAlgorithm::gridEdge const& ge3)
{
    xl = min(ge0.xl, min(ge1.xl, min(ge2.xl, ge3.xl)));
    xr = max(ge0.xr, max(ge1.xr, max(ge2.xr, ge3.xr)));

    if(xl > xr)
        xl = xr;
}

__device__
uchar calcCubeCase(
    uchar const& ec0, uchar const& ec1,
    uchar const& ec2, uchar const& ec3)
{
    // ec0 | (_,j,k)
    // ec1 | (_,j+1,k)
    // ec2 | (_,j,k+1)
    // ec3 | (_,j+1,k+1)

    uchar caseId = 0;
    if((ec0 == 0) || (ec0 == 2)) // 0 | (i,j,k)
        caseId |= 1;
    if((ec0 == 0) || (ec0 == 1)) // 1 | (i+1,j,k)
        caseId |= 2;
    if((ec1 == 0) || (ec1 == 1)) // 2 | (i+1,j+1,k)
        caseId |= 4;
    if((ec1 == 0) || (ec1 == 2)) // 3 | (i,j+1,k)
        caseId |= 8;
    if((ec2 == 0) || (ec2 == 2)) // 4 | (i,j,k+1)
        caseId |= 16;
    if((ec2 == 0) || (ec2 == 1)) // 5 | (i+1,j,k+1)
        caseId |= 32;
    if((ec3 == 0) || (ec3 == 1)) // 6 | (i+1,j+1,k+1)
        caseId |= 64;
    if((ec3 == 0) || (ec3 == 2)) // 7 | (i,j+1,k+1)
        caseId |= 128;
    return caseId;
}

__global__
void pass2gpu_cubeCases(
    int nx, int ny, int nz,
    uchar* edgeCases,
    FlyingEdgesAlgorithm::gridEdge* gridEdges,
    int* triCounter,
    uchar* cubeCases)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if(j >= ny-1 || k >= nz-1)
        return;

    FlyingEdgesAlgorithm::gridEdge& ge0 = gridEdges[k*ny + j];
    FlyingEdgesAlgorithm::gridEdge& ge1 = gridEdges[k*ny + j + 1];
    FlyingEdgesAlgorithm::gridEdge& ge2 = gridEdges[(k+1)*ny + j];
    FlyingEdgesAlgorithm::gridEdge& ge3 = gridEdges[(k+1)*ny + j + 1];

    uchar* ec0 = edgeCases + k*ny*(nx-1) + j*(nx-1);
    uchar* ec1 = edgeCases + k*ny*(nx-1) + (j+1)*(nx-1);
    uchar* ec2 = edgeCases + (k+1)*ny*(nx-1) + j*(nx-1);
    uchar* ec3 = edgeCases + (k+1)*ny*(nx-1) + (j+1)*(nx-1);

    int xl, xr;
    calcTrimValues(xl, xr, ge0, ge1, ge2, ge3);

    int triCount = 0;
    uchar* curCubeCases = cubeCases + k*(nx-1)*(ny-1) + j*(nx-1);

    int xstart = 0;
    int ystart = 0;
    int zstart = 0; // TODO don't set initial values in gridEdge Constructor;

    const bool* isCut;
    for(int i = xl; i != xr; ++i) // What happens here on a gpu?
                                  // I imagine it takes the max xr-xl of all blocks
    {
        uchar caseId = calcCubeCase(ec0[i], ec1[i], ec2[i], ec3[i]);

        curCubeCases[i] = caseId;

        // Can't imagine this would do anything on a gpu unless all threads
        // on a block evaluated to the same value.
        if(caseId == 0 || caseId == 255)
        {
            continue;
        }


        triCount += cuda_util::numTris[caseId];
        isCut = cuda_util::isCut[caseId]; // if xr == nx-1, then xr-1 is cut
                                          // so this will be set

        xstart += isCut[0];
        ystart += isCut[3];
        zstart += isCut[8];
    }

    triCounter[k*(ny-1) + j] = triCount;

    if(xr == nx-1)
    {
        // isCut was set at i = xr-1
        ystart += isCut[1];
        zstart += isCut[9];
    }

    ge0.xstart = xstart;
    ge0.ystart = ystart;
    ge0.zstart = zstart;
}

__global__
void pass2gpu_ghost_xz(
    int nx, int ny, int nz,
    uchar* edgeCases,
    FlyingEdgesAlgorithm::gridEdge* gridEdges)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if(k >= nz) // This function will deal with gridEdge at (_, ny-1, nz-1)
        return;

    bool isCorner = k == nz-1;

    int j = ny-1;

    FlyingEdgesAlgorithm::gridEdge& ge0 = gridEdges[k*ny + j];
    // If isCorner, this is just bogus.
    FlyingEdgesAlgorithm::gridEdge& ge1 = gridEdges[(1-isCorner)*(k+1)*ny + j];

    uchar* ec0 = edgeCases + k*ny*(nx-1) + j*(nx-1);
    // If isCorner, this is just bogus
    uchar* ec1 = edgeCases + (1-isCorner)*(k+1)*ny*(nx-1) + j*(nx-1);

    int xl = min(ge0.xl, nx*isCorner + (1-isCorner)*ge1.xl);
    int xr = max(ge0.xr, (1-isCorner)*ge1.xr);

    int xstart = 0;
    int zstart = 0; // TODO don't set initial values in gridEdge Constructor;

    uchar c0;
    uchar c1;

    for(int i = xl; i != xr; ++i)
    {
        c0 = ec0[i];
        c1 = ec1[i];

        // see if the edges are cut
        xstart += (c0 == 1 || c0 == 2);

        // bogus if isCorner
        zstart += ( (c0 == 0 && c1 == 1) || (c0 == 0 && c1 == 3) ||
                    (c0 == 1 && c1 == 2) || (c0 == 2 && c1 == 3) );
    }

    if(xr == nx-1)
    {
        // bogus if isCorner
        zstart += ( (c0 == 0 && c1 == 2) || (c0 == 0 && c1 == 3) ||
                    (c0 == 1 && c1 == 2) || (c0 == 1 && c1 == 3) );
    }

    ge0.xstart = xstart;
    ge0.ystart = 0;
    ge0.zstart = zstart*(1-isCorner);
}

__global__
void pass2gpu_ghost_xy(
    int nx, int ny, int nz,
    uchar* edgeCases,
    FlyingEdgesAlgorithm::gridEdge* gridEdges)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(j >= ny-1)
        return;

    int k = nz-1;

    FlyingEdgesAlgorithm::gridEdge& ge0 = gridEdges[k*ny + j];
    FlyingEdgesAlgorithm::gridEdge& ge1 = gridEdges[k*ny + j + 1];

    uchar* ec0 = edgeCases + k*ny*(nx-1) + j*(nx-1);
    uchar* ec1 = edgeCases + k*ny*(nx-1) + (j+1)*(nx-1);

    int xl = min(ge0.xl, ge1.xl);
    int xr = max(ge0.xr, ge1.xr);

    if(xl >= xr)
        return;

    int xstart = 0;
    int ystart = 0; // TODO don't set initial values in gridEdge Constructor;

    uchar c0;
    uchar c1;

    for(int i = xl; i != xr; ++i)
    {
        c0 = ec0[i];
        c1 = ec1[i];

        // see if the edges are cut
        xstart += (c0 == 1 || c0 == 2);
        ystart += ( (c0 == 0 && c1 == 1) || (c0 == 0 && c1 == 3) ||
                    (c0 == 1 && c1 == 2) || (c0 == 2 && c1 == 3) );
    }

    if(xr == nx-1)
    {
        ystart += ( (c0 == 0 && c1 == 2) || (c0 == 0 && c1 == 3) ||
                    (c0 == 1 && c1 == 2) || (c0 == 1 && c1 == 3) );
    }

    ge0.xstart = xstart;
    ge0.ystart = ystart;
    ge0.zstart = 0;
}

void FlyingEdgesAlgorithm::pass2()
{
    // pass2 calculates
    //   1) cubeCases for each block ray
    //   2) triCount for each block ray
    //   3) edgeRay count

    // 1st kernel: Calculate the 0, 1, 2 edge ray, cube cases, tricount
    // 2nd kernel: Calculate lost edges

    int ty = FE_BLOCK_WIDTH_Y;
    int tz = FE_BLOCK_WIDTH_Z;
    uint3 gridDim = make_uint3(((ny-1) + ty - 1) / ty, ((nz-1) + tz - 1) / tz, 1);
    uint3 blockDim = make_uint3(ty, tz, 1);

    if(!validKernelSize(gridDim, blockDim))
        std::cout << "GAHHHHHHHHHHHH GP2 ENGINE " << __LINE__ << std::endl; // TODO

    pass2gpu_cubeCases<<<gridDim, blockDim>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges,   // modified
        triCounter,  // modified
        cubeCases);  // modified

    // POSSIBLE to do this here TODO
    // hipFree(edgeCases);

    if(DEBUG)
    {
       std::cout << "MEOWWWWWW " << hipGetErrorString(hipGetLastError()) << std::endl;
    }

    if(DEBUG)
    {
        size_t sz = (nx-1)*(ny-1)*(nz-1)*sizeof(uchar);

        hipDeviceSynchronize();
        uchar* hostCubeCases = (uchar*)malloc(sz);
        hipMemcpy(hostCubeCases, cubeCases,
                   sz, hipMemcpyDeviceToHost);

        int count = 0;
        // TODO hostCubeCases is not the same every time.
        for(int i = 0; i != (nx-1)*(ny-1)*(nz-1); ++i)
        {
            if(hostCubeCases[i] != 0 && hostCubeCases[i] != 255)
                count += hostCubeCases[i];
        }
        std::cout << "Count cube cases " << count << std::endl;

        free(hostCubeCases);
    }

    // TODO these can be launched and executed independently of each other
    int bw = FE_BLOCK_WIDTH;

    // Making sure that the xz face takes care of the (_, ny-1, nz-1) gridEdge
    // BE CAREFUL. xz takes care of corner. don't use (nz-1)
    pass2gpu_ghost_xz<<<(nz + bw - 1) / bw, bw>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges);
    pass2gpu_ghost_xy<<<((ny-1) + bw - 1) / bw, bw>>>(
        nx, ny, nz,
        edgeCases,
        gridEdges);

    hipDeviceSynchronize();
std::cout << "MEOWWWWWW " << hipGetErrorString(hipGetLastError()) << std::endl;

    if(DEBUG)
    {
        size_t sz_ge = nx*ny*sizeof(gridEdge);
        gridEdge* hostges = (gridEdge*)malloc(sz_ge);
        auto w = hipMemcpy(hostges, gridEdges,
                   sz_ge, hipMemcpyDeviceToHost);
        if(w != hipSuccess)
        {
            std::cout << "GHASDCFAKSCLKASCKAS:CKASL:CKAS:DLCKASD:" << std::endl;
            std::cout << hipGetErrorString(w) << std::endl;
        }

        int sumxstart = 0;
        for(int idx = 0; idx != nx*ny; ++idx)
        {
            sumxstart += hostges[idx].xstart;
        }
        std::cout << "sumxstart " << sumxstart << std::endl;
        free(hostges);
    }

}

///////////////////////////////////////////////////////////////////////////////
// Pass 3 of the algorithm
///////////////////////////////////////////////////////////////////////////////

__global__
void pass3gpu_blockAccum(
    int nx, int ny, int nz, // which are needed TODO?
    int* triCounter,
    FlyingEdgesAlgorithm::gridEdge* gridEdges,
    int* blockAccum)
{
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    // step 1: accumulate individual y thread
    // step 2: calc block sum
    // step 3: __syncthreads
    // step 4: add to individual y thread

    __shared__ int accum[4*FE_BLOCK_WIDTH];

    if(k < nz)
    {
        int tmp;
        int accumX   = 0;
        int accumY   = 0;
        int accumZ   = 0;
        int accumTri = 0;
        for(int j = 0; j != ny; ++j)
        {
            FlyingEdgesAlgorithm::gridEdge& ge = gridEdges[k*ny + j];

            tmp = ge.xstart;
            ge.xstart = accumX;
            accumX += tmp;

            tmp = ge.ystart;
            ge.ystart = accumY;
            accumY += tmp;

            tmp = ge.zstart;
            ge.zstart = accumZ;
            accumZ += tmp;
        }

        if(k < nz-1)
        {
            for(int j = 0; j != ny-1; ++j)
            {
                int& curTriCount = triCounter[k*(ny-1) + j];

                tmp = curTriCount;
                curTriCount = accumTri;
                accumTri += tmp;
            }
        }

        accum[4*threadIdx.y + 0] = accumX;
        accum[4*threadIdx.y + 1] = accumY;
        accum[4*threadIdx.y + 2] = accumZ;
        accum[4*threadIdx.y + 3] = accumTri;
    }

    __syncthreads();

    if(k < nz)
    {
        if(threadIdx.y == 0) // agh!
        {
            for(int idx = 1; idx != blockDim.y; ++idx)
            {
                accum[4*idx + 0] += accum[4*(idx-1) + 0];
                accum[4*idx + 1] += accum[4*(idx-1) + 1];
                accum[4*idx + 2] += accum[4*(idx-1) + 2];
                accum[4*idx + 3] += accum[4*(idx-1) + 3];
            }

            // answer for global accumulation
            blockAccum[4*blockIdx.y + 0] = accum[4*(blockDim.y-1) + 0];
            blockAccum[4*blockIdx.y + 1] =  accum[4*(blockDim.y-1) + 1];
            blockAccum[4*blockIdx.y + 2] =  accum[4*(blockDim.y-1) + 2];
            blockAccum[4*blockIdx.y + 3] = accum[4*(blockDim.y-1) + 3];
        }
    }
    __syncthreads();

    if(threadIdx.y == 0 || k >= nz)
        return;

    bool isEndK = k == nz-1;
    for(int j = 0; j != ny-1; ++j)
    {
        FlyingEdgesAlgorithm::gridEdge& ge = gridEdges[k*ny + j];

        ge.xstart += accum[4*(threadIdx.y-1) + 0];
        ge.ystart += accum[4*(threadIdx.y-1) + 1];
        ge.zstart += accum[4*(threadIdx.y-1) + 2];

        // put z stuff here..
        if(!isEndK)
            triCounter[k*(ny-1) + j] = accum[4*(threadIdx.y-1) + 3];
    }

    FlyingEdgesAlgorithm::gridEdge& ge = gridEdges[k*ny + (ny-1)];
    ge.xstart += accum[4*(threadIdx.y-1) + 0];
    ge.ystart += accum[4*(threadIdx.y-1) + 1];
    ge.zstart += accum[4*(threadIdx.y-1) + 2];
}

__global__ // TODO can split up along j here easy enough.
void pass3gpu_gridAccum(
    int nx, int ny, int nz, // which are needed TODO?
    int* triCounter,
    FlyingEdgesAlgorithm::gridEdge* gridEdges,
    int* blockAccum) // used as input here
{
    // not adding to the first block!
    //
    // add to individual y threads
    int k = (blockIdx.z + 1)*blockDim.z + threadIdx.z;

    if (k >= nz)
        return;

    int addX   = blockAccum[4*blockIdx.z + 0];
    int addY   = blockAccum[4*blockIdx.z + 1];
    int addZ   = blockAccum[4*blockIdx.z + 2];
    int addTri = blockAccum[4*blockIdx.z + 3];

    for(int j = 0; j != ny; ++j)
    {
        FlyingEdgesAlgorithm::gridEdge& ge = gridEdges[k*ny + j];
        ge.xstart += addX;
        ge.ystart += addY;
        ge.zstart += addZ;
    }

    if(k >= nz-1)
        return;

    for(int j = 0; j != ny-1; ++j)
    {
        triCounter[k*(ny-1) + j] += addTri;
    }
}

// Can make prettier?
void FlyingEdgesAlgorithm::pass3()
{
    // Split the z axis
    // Kernel 1: calculate the accum values on block sync
    //           then accum individual values
    // Use that info accum each block (except the first one)
    // Kernel 2: just add values to individual threads
    int tz = FE_BLOCK_WIDTH;

    int numBlocks = (nz + tz - 1) / tz;

    // there are four because: xstart, ystart, zstart, triaccum
    int sizeBlocks = 4 * numBlocks * sizeof(int);

    uint3 gridDim = make_uint3(1, numBlocks, 1);
    uint3 blockDim = make_uint3(1, tz, 1);

    int* hostBlockAccum = (int*)malloc(sizeBlocks);
    for(int idx = 0; idx != 4*numBlocks; ++idx)
    {
        hostBlockAccum[idx] = 0;
    }

    int* deviceBlockAccum;
    hipMalloc(&deviceBlockAccum, sizeBlocks);

    hipMemcpy(deviceBlockAccum, hostBlockAccum,
                   sizeBlocks, hipMemcpyHostToDevice);

    // Accumulate values locally

    if(!validKernelSize(gridDim, blockDim))
        std::cout << "GAHHHHHHHHHHHH GP2 ENGINE " << __LINE__ << std::endl; // TODO


    pass3gpu_blockAccum<<<gridDim, blockDim>>>(
        nx, ny, nz,
        triCounter,
        gridEdges,
        deviceBlockAccum);

    hipMemcpy(hostBlockAccum, deviceBlockAccum,
               sizeBlocks, hipMemcpyDeviceToHost);

    if(DEBUG)
    {
        std::cout << "ACCUM ";
        for(int idx = 0; idx != 4*numBlocks; ++idx)
        {
            std::cout << hostBlockAccum[idx] << " ";
        }
        std::cout << std::endl;

        hipDeviceSynchronize();

        std::cout << "MEOWWWWWW " << hipGetErrorString(hipGetLastError()) << std::endl;
    }

    if(numBlocks != 1)
    {
        // std::partial_sum(2 2 3 4  3  2  2 ) TODO not using it get rid of header
        // goes to         (2 4 7 11 14 16 18)
        // std::partial_sum(hostBlockAccum, hostBlockAccum + numBlocks, hostBlockAccum);

        for(int i = 4; i != 4*numBlocks; i += 4)
        {
            hostBlockAccum[i+0] += hostBlockAccum[i-4];
            hostBlockAccum[i+1] += hostBlockAccum[i-3];
            hostBlockAccum[i+2] += hostBlockAccum[i-2];
            hostBlockAccum[i+3] += hostBlockAccum[i-1];
        }
        // note: the last values in hostBlockAccum should contain total counts

        // The first block is done so it is ignored
        // and the last info in BlockAccum isn't needed (its the total counts)
        hipMemcpy(deviceBlockAccum, hostBlockAccum,
                   sizeBlocks - 4 * sizeof(int), hipMemcpyHostToDevice);

        // TODO
        if(!validKernelSize(gridDim, blockDim))
            std::cout << "GAHHHHHHHHHHHH GP2 ENGINE " << __LINE__ << std::endl; // TODO

        // Accumulate values from other blocks
        gridDim = make_uint3(1, 1, numBlocks - 1);
        pass3gpu_gridAccum<<<gridDim, blockDim>>>(
            nx, ny, nz,
            triCounter,
            gridEdges,
            deviceBlockAccum);
    }

    // Allocate memory for points, normals and tris
    outputAllocated = true;
    numPoints = hostBlockAccum[4*(numBlocks-1) + 0] +
                hostBlockAccum[4*(numBlocks-1) + 1] +
                hostBlockAccum[4*(numBlocks-1) + 2];
    numTris   = hostBlockAccum[4*(numBlocks-1) + 3];

//    hipMalloc(&points,  3*sizeof(scalar_t)*numPoints);
//    hipMalloc(&normals, 3*sizeof(scalar_t)*numPoints);
//    hipMalloc(&tris, 3*sizeof(int)*numTris);

    if(DEBUG)
    {
        std::cout << "numpoints" << numPoints << std::endl;
        std::cout << "numtris"   << numTris   << std::endl;
    }

    // free memory used in this function
    free(hostBlockAccum);
    hipFree(deviceBlockAccum);

    hipDeviceSynchronize();

    if(DEBUG)
    {
        std::cout << "MEOWWWWWW " << hipGetErrorString(hipGetLastError()) << std::endl;
    }
}

///////////////////////////////////////////////////////////////////////////////
// Pass 4 of the algorithm
///////////////////////////////////////////////////////////////////////////////
__device__
void computeGradient(
    int const& i, int const& j, int const& k,
    int const& nx, int const& ny, int const& nz,
    scalar_t* data,
    scalar_t* spacing,
    scalar_t* point)
{
    scalar_t x0[2];
    scalar_t x1[2];
    scalar_t x2[2];
    scalar_t run[3];

    size_t dataIdx = k*nx*ny + j*nx + i;

    if (i == 0)
    {
        x0[0] = data[dataIdx + 1];
        x0[1] = data[dataIdx];
        run[0] = spacing[0];
    }
    else if (i == (nx - 1))
    {
        x0[0] = data[dataIdx];
        x0[1] = data[dataIdx - 1];
        run[0] = spacing[0];
    }
    else
    {
        x0[0] = data[dataIdx + 1];
        x0[1] = data[dataIdx - 1];
        run[0] = 2 * spacing[0];
    }

    if (j == 0)
    {
        x1[0] = data[dataIdx + nx];
        x1[1] = data[dataIdx];
        run[1] = spacing[1];
    }
    else if (j == (ny - 1))
    {
        x1[0] = data[dataIdx];
        x1[1] = data[dataIdx - nx];
        run[1] = spacing[1];
    }
    else
    {
        x1[0] = data[dataIdx + nx];
        x1[1] = data[dataIdx - ny];
        run[1] = 2 * spacing[1];
    }

    if (k == 0)
    {
        x2[0] = data[dataIdx + nx*ny];
        x2[1] = data[dataIdx];
        run[2] = spacing[2];
    }
    else if (k == (nz - 1))
    {
        x2[0] = data[dataIdx];
        x2[1] = data[dataIdx - nx*ny];
        run[2] = spacing[2];
    }
    else
    {
        x2[0] = data[dataIdx + nx*ny];
        x2[1] = data[dataIdx - nx*ny];
        run[2] = 2 * spacing[2];
    }

    point[0] = (x0[1] - x0[0]) / run[0];
    point[1] = (x1[1] - x1[0]) / run[1];
    point[2] = (x2[1] - x2[0]) / run[2];
}


__device__
void getCubeInfo(
    int i, int j, int k,
    int nx, int ny, int nz,
    scalar_t* pointValues, scalar_t* zeroPos, scalar_t* spacing,
    scalar_t* pointCube, scalar_t* isovalCube, scalar_t* gradCube)
{
    isovalCube[0] = pointValues[k*ny*nx + j*nx + i];
    isovalCube[1] = pointValues[k*ny*nx + j*nx + i+1];
    isovalCube[2] = pointValues[k*ny*nx + (j+1)*nx + i+1];
    isovalCube[3] = pointValues[k*ny*nx + (j+1)*nx + i];
    isovalCube[4] = pointValues[(k+1)*ny*nx + j*nx + i];
    isovalCube[5] = pointValues[(k+1)*ny*nx + j*nx + i+1];
    isovalCube[6] = pointValues[(k+1)*ny*nx + (j+1)*nx + (i+1)];
    isovalCube[7] = pointValues[(k+1)*ny*nx + (j+1)*nx + i];

    scalar_t xpos = zeroPos[0] + i * spacing[0];
    scalar_t ypos = zeroPos[1] + j * spacing[1];
    scalar_t zpos = zeroPos[2] + k * spacing[2];

    pointCube[0*3 + 0] = xpos;
    pointCube[0*3 + 1] = ypos;
    pointCube[0*3 + 2] = zpos;

    pointCube[1*3 + 0] = xpos + spacing[0];
    pointCube[1*3 + 1] = ypos;
    pointCube[1*3 + 2] = zpos;

    pointCube[2*3 + 0] = xpos + spacing[0];
    pointCube[2*3 + 1] = ypos + spacing[1];
    pointCube[2*3 + 2] = zpos;

    pointCube[3*3 + 0] = xpos;
    pointCube[3*3 + 1] = ypos + spacing[1];
    pointCube[3*3 + 2] = zpos;

    pointCube[4*3 + 0] = xpos;
    pointCube[4*3 + 1] = ypos;
    pointCube[4*3 + 2] = zpos + spacing[2];

    pointCube[5*3 + 0] = xpos + spacing[0];
    pointCube[5*3 + 1] = ypos;
    pointCube[5*3 + 2] = zpos + spacing[2];

    pointCube[6*3 + 0] = xpos + spacing[0];
    pointCube[6*3 + 1] = ypos + spacing[1];
    pointCube[6*3 + 2] = zpos + spacing[2];

    pointCube[7*3 + 0] = xpos;
    pointCube[7*3 + 1] = ypos + spacing[1];
    pointCube[7*3 + 2] = zpos + spacing[2];

    computeGradient(i  , j  , k  , nx, ny, nz, pointValues, spacing, gradCube + 3*0);
    computeGradient(i+1, j  , k  , nx, ny, nz, pointValues, spacing, gradCube + 3*1);
    computeGradient(i+1, j+1, k  , nx, ny, nz, pointValues, spacing, gradCube + 3*2);
    computeGradient(i  , j+1, k  , nx, ny, nz, pointValues, spacing, gradCube + 3*3);
    computeGradient(i  , j  , k+1, nx, ny, nz, pointValues, spacing, gradCube + 3*4);
    computeGradient(i+1, j  , k+1, nx, ny, nz, pointValues, spacing, gradCube + 3*5);
    computeGradient(i+1, j+1, k+1, nx, ny, nz, pointValues, spacing, gradCube + 3*6);
    computeGradient(i  , j+1, k+1, nx, ny, nz, pointValues, spacing, gradCube + 3*7);
}

__device__
void interpolate(
    scalar_t const& weight,
    scalar_t* a,
    scalar_t* b,
    scalar_t* out)
{
    out[0] = a[0] + (weight * (b[0] - a[0]));
    out[1] = a[1] + (weight * (b[1] - a[1]));
    out[2] = a[2] + (weight * (b[2] - a[2]));
}

__device__
void interpolateOnCube(
    uchar const& edge,
    scalar_t const& isoval,
    scalar_t* pts,
    scalar_t* isovals,
    scalar_t* out)
{
    uchar i0 = cuda_util::edgeVertices[edge][0];
    uchar i1 = cuda_util::edgeVertices[edge][1];

    scalar_t weight = (isoval - isovals[i0]) / (isovals[i1] - isovals[i0]);
    interpolate(weight, pts + 3*i0, pts + 3*i1, out);
}

__global__
void pass4gpu_pointsAndNormals(
    int nx, int ny, int nz,
    scalar_t* pointValues, scalar_t* zeroPos, scalar_t* spacing,
    scalar_t isoval,
    FlyingEdgesAlgorithm::gridEdge* gridEdges,
    int* triCounter,
    uchar* cubeCases,
    scalar_t* points, scalar_t* normals, int* tris)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if(DEBUG)
    {
        if(j == 0 && k == 0)
        {
//            for(int i = 0; i != 3*1370424; ++i)
//            {
//                points[i] = -1;
//                normals[i] = -1;
//            }

            for(int i = 0; i != 3*2740864; ++i)
                tris[i] = -1;
        }
    }

    if(j >= ny-1 || k >= nz-1)
        return;

    FlyingEdgesAlgorithm::gridEdge& ge0 = gridEdges[k*ny + j];
    FlyingEdgesAlgorithm::gridEdge& ge1 = gridEdges[k*ny + j+1];
    FlyingEdgesAlgorithm::gridEdge& ge2 = gridEdges[(k+1)*ny + j];
    FlyingEdgesAlgorithm::gridEdge& ge3 = gridEdges[(k+1)*ny + j+1];

    int xl, xr;
    calcTrimValues(xl, xr, ge0, ge1, ge2, ge3);

    if(xl == xr)
        return;

    size_t triIdx = triCounter[k*(ny-1) + j];
    uchar* curCubeCaseIds = cubeCases + (nx-1)*(k*(ny-1) + j);

    size_t x0counter = 0;
    size_t y0counter = 0;
    size_t z0counter = 0;

    size_t x1counter = 0;
    size_t z1counter = 0;

    size_t x2counter = 0;
    size_t y2counter = 0;

    size_t x3counter = 0;

    bool isYEnd = (j == ny-2);
    bool isZEnd = (k == nz-2);

    scalar_t pointCube[8*3];
    scalar_t isovalCube[8];
    scalar_t gradCube[8*3];

    for(size_t i = xl; i != xr; ++i)
    {
        bool isXEnd = (i == nx-2);

        uchar caseId = curCubeCaseIds[i];

        if(caseId == 0 || caseId == 255)
        {
            continue;
        }

        const bool* isCut = cuda_util::isCut[caseId]; // has 12 elements

        // Most of the information contained in pointCube, isovalCube
        // and gradCube will be used--but not necessarily all. It has
        // not been tested whether or not obtaining only the information
        // needed will provide a significant speedup--but
        // most likely not.

        // fill out pointCube, isovalCube and gradCube
        getCubeInfo(i, j, k,
                    nx, ny, nz,
                    pointValues, zeroPos, spacing,
                    pointCube, isovalCube, gradCube);

        // Add Points and normals.
        // Calculate global indices for triangles
        int globalIdxs[12];
        if(isCut[0])
        {
            int idx = ge0.xstart + x0counter;
            interpolateOnCube(0, isoval, pointCube, isovalCube, points + 3*idx);
            interpolateOnCube(0, isoval, gradCube, isovalCube, normals + 3*idx);
            globalIdxs[0] = idx;
            ++x0counter;
        }

        if(isCut[3])
        {
            int idx = ge0.ystart + y0counter;
            interpolateOnCube(3, isoval, pointCube, isovalCube, points + 3*idx);
            interpolateOnCube(3, isoval, gradCube, isovalCube, normals + 3*idx);
            globalIdxs[3] = idx;
            ++y0counter;
        }

        if(isCut[8])
        {
            int idx = ge0.zstart + z0counter;
            interpolateOnCube(8, isoval, pointCube, isovalCube, points + 3*idx);
            interpolateOnCube(8, isoval, gradCube, isovalCube, normals + 3*idx);
            globalIdxs[8] = idx;
            ++z0counter;
        }

        // Note:
        //   e1, e5, e9 and e11 will be visited in the next iteration
        //   when they are e3, e7, e8 and 10 respectively. So don't
        //   increment their counters. When the cube is an edge cube,
        //   their counters don't need to be incremented because they
        //   won't be used agin.

        // Manage boundary cases if needed. Otherwise just update
        // globalIdx.
        if(isCut[1])
        {
            int idx = ge0.ystart + y0counter;
            if(isXEnd)
            {
                interpolateOnCube(1, isoval, pointCube, isovalCube, points + 3*idx);
                interpolateOnCube(1, isoval, gradCube, isovalCube, normals + 3*idx);
                // y0counter counter doesn't need to be incremented
                // because it won't be used again.
            }
            globalIdxs[1] = idx;
        }

        if(isCut[9])
        {
            int idx = ge0.zstart + z0counter;
            if(isXEnd)
            {
                interpolateOnCube(9, isoval, pointCube, isovalCube, points + 3*idx);
                interpolateOnCube(9, isoval, gradCube, isovalCube, normals + 3*idx);
                // z0counter doesn't need to in incremented.
            }
            globalIdxs[9] = idx;
        }

        if(isCut[2])
        {
            int idx = ge1.xstart + x1counter;
            if(isYEnd)
            {
                interpolateOnCube(2, isoval, pointCube, isovalCube, points + 3*idx);
                interpolateOnCube(2, isoval, gradCube, isovalCube, normals + 3*idx);
            }
            globalIdxs[2] = idx;
            ++x1counter;
        }

        if(isCut[10])
        {
            int idx = ge1.zstart + z1counter;
            if(isYEnd)
            {
                interpolateOnCube(10, isoval, pointCube, isovalCube, points + 3*idx);
                interpolateOnCube(10, isoval, gradCube, isovalCube, normals + 3*idx);
            }
            globalIdxs[10] = idx;
            ++z1counter;
        }

        if(isCut[4])
        {
            int idx = ge2.xstart + x2counter;
            if(isZEnd)
            {
                interpolateOnCube(4, isoval, pointCube, isovalCube, points + 3*idx);
                interpolateOnCube(4, isoval, gradCube, isovalCube, normals + 3*idx);
            }
            globalIdxs[4] = idx;
            ++x2counter;
        }

        if(isCut[7])
        {
            int idx = ge2.ystart + y2counter;
            if(isZEnd)
            {
                interpolateOnCube(7, isoval, pointCube, isovalCube, points + 3*idx);
                interpolateOnCube(7, isoval, gradCube, isovalCube, normals + 3*idx);
            }
            globalIdxs[7] = idx;
            ++y2counter;
        }

        if(isCut[11])
        {
            int idx = ge1.zstart + z1counter;
            if(isXEnd and isYEnd)
            {
                interpolateOnCube(11, isoval, pointCube, isovalCube, points + 3*idx);
                interpolateOnCube(11, isoval, gradCube, isovalCube, normals + 3*idx);
                // z1counter does not need to be incremented.
            }
            globalIdxs[11] = idx;
        }

        if(isCut[5])
        {
            int idx = ge2.ystart + y2counter;
            if(isXEnd and isZEnd)
            {
                interpolateOnCube(5, isoval, pointCube, isovalCube, points + 3*idx);
                interpolateOnCube(5, isoval, gradCube, isovalCube, normals + 3*idx);
                // y2 counter does not need to be incremented.
            }
            globalIdxs[5] = idx;
        }

        if(isCut[6])
        {
            int idx = ge3.xstart + x3counter;
            if(isYEnd and isZEnd)
            {
                interpolateOnCube(6, isoval, pointCube, isovalCube, points + 3*idx);
                interpolateOnCube(6, isoval, gradCube, isovalCube, normals + 3*idx);
            }
            globalIdxs[6] = idx;
            ++x3counter;
        }

        // Add triangles
        const char* caseTri = cuda_util::caseTriangles[caseId]; // size 16
        for(int idx = 0; caseTri[idx] != -1; idx += 3)
        {
            tris[3*triIdx + 0] = i;
            tris[3*triIdx + 1] = j;
            tris[3*triIdx + 2] = k;

//            tris[3*triIdx + 0] = globalIdxs[caseTri[idx]];
//            tris[3*triIdx + 1] = globalIdxs[caseTri[idx+1]];
//            tris[3*triIdx + 2] = globalIdxs[caseTri[idx+2]];
//            ++triIdx;
        }
    }
}


void FlyingEdgesAlgorithm::pass4()
{
    // pass4 calculates points and normals
    //   1) points and normals

    // 1st kernel:           Calculate the main cube rays
    // 2nd and third kernel:

    int ty = 1;//FE_BLOCK_WIDTH_Y / 2; // divide by 2? TODO figure out this problem..
    int tz = 1;//FE_BLOCK_WIDTH_Z / 2; // gah....
    uint3 gridDim = make_uint3(((ny-1) + ty - 1) / ty, ((nz-1) + tz - 1) / tz, 1);
    uint3 blockDim = make_uint3(ty, tz, 1);

    std::cout << gridDim.x << ", " << gridDim.y << ", " << gridDim.z << std::endl;
    std::cout << blockDim.x << ", " << blockDim.y << ", " << blockDim.z << std::endl;

    if(!validKernelSize(gridDim, blockDim))
        std::cout << "GAHHHHHHHHHHHH GP2 ENGINE " << __LINE__ << std::endl; // TODO

    if(DEBUG)
    {
       hipDeviceSynchronize();
    }

    pass4gpu_pointsAndNormals<<<gridDim, blockDim>>>(
        nx, ny, nz,                                    // input
        pointValues, zeroPos, spacing,                 // input
        isoval,                                        // input
        gridEdges, triCounter, cubeCases,              // input
        points, normals, tris);                        // output

    if(DEBUG)
    {
       hipDeviceSynchronize();
       std::cout << "MEOWWWWWW " << hipGetErrorString(hipGetLastError()) << std::endl;
    }

    if(DEBUG)
    {
        size_t sz = 3 * numPoints * sizeof(scalar_t);

        scalar_t* hostPts = (scalar_t*)malloc(sz);
        scalar_t* hostNrs = (scalar_t*)malloc(sz);
        int*      hostTrs = (int*)malloc(3*numTris*sizeof(int));

        hipMemcpy(hostPts, points,  sz, hipMemcpyDeviceToHost);
        hipMemcpy(hostNrs, normals, sz, hipMemcpyDeviceToHost);
        hipMemcpy(hostTrs, tris, 3*numTris*sizeof(int), hipMemcpyDeviceToHost);

        scalar_t accumP = 0.0;
        for(int idx = 0; idx != 3 * numPoints; ++idx)
        {
            accumP += hostPts[idx];
            accumP += hostTrs[idx];

            while(accumP >= 1000000)
                accumP -= 1000000;
        }

        int accumT = 0;
        int num0 = -1;
        int num9 = 0;
        int num8 = 0;
        int num7 = 0;

        int numSetPoints = 0;
        for(int idx = 0; idx != 3 * numPoints; ++idx)
        {
            if(hostPts[idx] != -1)
                numSetPoints += 1;
        }

        std::cout << "numSetPoints " << numSetPoints << std::endl;

        for(int idx = 0; idx != 3 * numTris; ++idx)
        {
            if(hostTrs[idx] == 0)
                num0 += 1;

            if(hostTrs[idx] == 9)
                num9 += 1;

            if(hostTrs[idx] == 8)
                num8 += 1;

            if(hostTrs[idx] == 7)
                num7 += 1;

            accumT += hostTrs[idx];

            while(accumT >= 1000000)
                accumT -= 1000000;
        }

        std::cout << "pass 4 hashsum " << accumP << ", " << accumT << std::endl;
        std::cout << "num0 in Tris "   << num0 <<                     std::endl;
        std::cout << "num9 in Tris "   << num9 <<                     std::endl;
        std::cout << "num8 in Tris "   << num8 <<                     std::endl;
        std::cout << "num7 in Tris "   << num7 <<                     std::endl;

        for(int idx = 0; idx != numTris*3; idx += 3)
        {
            if(hostTrs[idx] != -1)
            {
                std::cout << hostTrs[idx+0] << ", "
                          << hostTrs[idx+1] << ", "
                          << hostTrs[idx+2] << std::endl;
            }
        }

        free(hostPts);
        free(hostNrs);
        free(hostTrs);
    }
}


