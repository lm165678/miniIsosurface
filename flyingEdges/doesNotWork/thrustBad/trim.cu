
#include <hip/hip_runtime.h>
/*
    struct Trim
    {
        Trim(t3<const int> n)
          : left(n.y*n.z, n.x),
            right(n.y*n.z, 0),
            edgeCases((n.x-1)*n.y*n.z),
            iter_helper(edgeCases.begin(), n.y*n.z, n.x-1)
        {}

        thrust::device_vector<int> left;
        thrust::device_vector<int> right;
        thrust::device_vector<uchar> edgeCases;
        iter_access_helper<uchar> iter_helper;

        using reference_type =
            thrust::tuple<
                int,
                int,
                typename thrust::device_vector<uchar>::iterator>;

        using iterator_tuple =
            thrust::tuple<
                typename thrust::device_vector<int>::iterator,
                typename thrust::device_vector<int>::iterator,
                typename iter_access_helper<uchar>::iterator>;

        using iterator = thrust::zip_iterator<iterator_tuple>;

        iterator begin()
        {
            return thrust::make_zip_iterator(
                thrust::make_tuple(
                    left.begin(),
                    right.begin(),
                    iter_helper.begin()));
        }

        iterator end()
        {
            return thrust::make_zip_iterator(
                thrust::make_tuple(
                    left.end(),
                    right.end(),
                    iter_helper.end()));
        }
    };
*/







/*
    struct set_trim_values
      : public thrust::binary_function<
            typename Trim::reference_type,                      // arg 1
            typename thrust::device_vector<scalar_t>::iterator, // arg 2
            typename Trim::reference_type>                      // out
    {
        set_trim_values(
            scalar_t const& isoval,
            int const& nx)
          : isoval(isoval),
            nx(nx)
        {}

        // This will not work on host because of using
        // edgeCases from device_vector...
        __host__
        typename Trim::reference_type
        operator()(
            typename Trim::reference_type trim_values,
            typename thrust::device_vector<scalar_t>::iterator curPoints)
        {
            // use curPoints to set xl, xr, curEdgeCases
            int xl = thrust::get<0>(trim_values);
            int xr = thrust::get<1>(trim_values);

            using iterator = typename thrust::device_vector<uchar>::iterator;
            iterator edgeCases = thrust::get<2>(trim_values);

            // TODO set all of isGE at once.

            bool isGE[2];
            isGE[0] = (curPoints[0] >= isoval);
            for(int i = 1; i != nx; ++i)
            {
                isGE[i%2] = (curPoints[i] >= isoval);

//                edgeCases[i-1] = calcCaseEdge(isGE[(i+1)%2], isGE[i%2]);
//
                if(*(edgeCases + i-1) == 1 || *(edgeCases + i-1) == 2)
                {
                    if(xl > xr)
                    {
                        xl = i-1;
                    }
                    xr = i;
                }
            }

            return thrust::make_tuple(xl, xr, edgeCases);
        }

        scalar_t const& isoval;
        int const& nx;
    };

    void pass1()
    {
        thrust::transform(
            trim.begin(),       // input1
            trim.end(),
            image.ray_begin(),  // input2
            trim.begin(),       // output
            set_trim_values(    // binary function (*input1, *input2)
                isoval,
                n.x));
    }
*/

